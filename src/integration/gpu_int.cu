#include "hip/hip_runtime.h"
#include "gpu_int.cuh"

void resizeToFit(double4 *cpu, double4 *gpu1, double4 *gpu2, size_t &size, int particles) {

    size_t neededsize = particles * 2 * sizeof(double4);

    if(neededsize > size){
        while(neededsize > size){
            size *= 1.5;
        }

        checkGPUError(hipFree(gpu1));
        checkGPUError(hipFree(gpu2));
        checkGPUError(hipHostFree(cpu));

        checkGPUError(hipHostMalloc(&cpu, size));
        checkGPUError(hipMalloc(&gpu1, size));
        checkGPUError(hipMalloc(&gpu2, size));

    }


}