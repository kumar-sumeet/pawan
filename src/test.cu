#include "hip/hip_runtime.h"

#include "test.cuh"
#include <iostream>

#include "wake/wake.h"
#include "wake/test_wake.h"
#include <gsl/gsl_vector_double.h>
#include "interaction/interaction_utils.h"
#include "interaction/interaction_utils_gpu.cuh"
#include "interaction/interaction.h"
#include "interaction/gpu.cuh"
#include <gsl/gsl_rng.h>
#include "io/io.h"
#include "integration/gpu_euler.cuh"
#include "wake/ring.h"
#include "wake/vring.h"
#include "resolve/resolve.h"

using namespace pawan;

constexpr double epsilon = 1e-10;


//output errors with location
bool closeToZero(double gpuV);

#define checkGPUError(ans) checkGPUError_((ans), __FILE__, __LINE__)

__inline__ void checkGPUError_(hipError_t errorCode, const char* file, int line){

    if(errorCode != hipSuccess) {
        //report error and stop
        std::cout << "Cuda Error: " << hipGetErrorString(errorCode) << "\nin " << file << ", line " << line;
        exit(EXIT_FAILURE);
    }
}

__global__ void testKernel(double nu,
                           const double4 *data,
                           double3 *returnVals){
    INTERACT_GPU(nu, data[0], data[2], data[1], data[3], returnVals[0], returnVals[1]);
}



bool testSingleInteract(double nu, double s_src, double s_trg, gsl_vector *r_src, gsl_vector *r_trg, gsl_vector *a_src,
                        gsl_vector *a_trg, double v_src, double v_trg) {

    //Outputs of CPU version
    double vx_s = 0.0, vy_s = 0.0, vz_s = 0.0;
    double qx_s = 0.0, qy_s = 0.0, qz_s = 0.0;
    gsl_vector *dr_trg = gsl_vector_alloc(3);
    gsl_vector *da_trg = gsl_vector_alloc(3);    //these are the exact negative -> ignore


    INTERACT(nu, s_src, s_trg, r_src, r_trg, a_src, a_trg, v_src, v_trg, dr_trg, da_trg, vx_s, vy_s, vz_s, qx_s, qy_s, qz_s);

    //std::cout << "CPU velocity " << vx_s << ", " << vy_s << ", " << vz_s << "\n";
    // std::cout << "CPU velocity' " << gsl_vector_get(dr_trg,0) << ", " << gsl_vector_get(dr_trg,1)<< ", " << gsl_vector_get(dr_trg,2) << "\n";
    //std::cout << "CPU retvorticity " << qx_s << ", " << qy_s << ", " << qz_s << "\n";

    //copy values to the format used by the gpu function
    double4 *data;
    double3 *retVals;

    checkGPUError(hipMallocManaged(&data,4* sizeof(double4)));
    checkGPUError(hipMallocManaged(&retVals,2* sizeof(double3)));

    data[0].x = gsl_vector_get(r_src, 0);
    data[0].y = gsl_vector_get(r_src, 1);
    data[0].z = gsl_vector_get(r_src, 2);
    data[0].w = s_src;

    data[1].x = gsl_vector_get(a_src, 0);
    data[1].y = gsl_vector_get(a_src, 1);
    data[1].z = gsl_vector_get(a_src, 2);
    data[1].w = v_src;

    data[2].x = gsl_vector_get(r_trg, 0);
    data[2].y = gsl_vector_get(r_trg, 1);
    data[2].z = gsl_vector_get(r_trg, 2);
    data[2].w = s_trg;

    data[3].x = gsl_vector_get(a_trg, 0);
    data[3].y = gsl_vector_get(a_trg, 1);
    data[3].z = gsl_vector_get(a_trg, 2);
    data[3].w = v_trg;

    testKernel<<<1,1>>>(nu,data,retVals);
    checkGPUError(hipDeviceSynchronize());

    //std::cout << "GPU velocity " << retVals[0].x << ", " << retVals[0].y << ", " << retVals[0].z << "\n";
    //std::cout << "GPU retvorticity " << retVals[1].x << ", " << retVals[1].y << ", " << retVals[1].z << "\n";


    bool equal = gsl_fcmp(retVals[0].x, vx_s, epsilon) == 0
                 && gsl_fcmp(retVals[0].y, vy_s, epsilon) == 0
                 && gsl_fcmp(retVals[0].z, vz_s, epsilon) == 0
                 && gsl_fcmp(retVals[1].x, qx_s, epsilon) == 0
                 && gsl_fcmp(retVals[1].y, qy_s, epsilon) == 0
                 && gsl_fcmp(retVals[1].z, qz_s, epsilon) == 0;

    checkGPUError(hipFree(data));
    checkGPUError(hipFree(retVals));
    gsl_vector_free(dr_trg);
    gsl_vector_free(da_trg);

    return equal;
}


void testInteractWithRandomValues() {
    gsl_vector *r_src = gsl_vector_alloc(3); //Position
    gsl_vector *a_src = gsl_vector_alloc(3); //vorticity
    gsl_vector *r_trg = gsl_vector_alloc(3);
    gsl_vector *a_trg = gsl_vector_alloc(3);

    gsl_rng * r;
    const gsl_rng_type * T;

    gsl_rng_env_setup();
    T = gsl_rng_default;
    r = gsl_rng_alloc (T);

    int wrong = 0;
    int iterations = 1000000;

    for(int i = 0; i < iterations; i++) {

        double nu = 2.5e-3;

        gsl_vector_set(r_trg, 0, gsl_rng_uniform(r) * 6 -3);
        gsl_vector_set(r_trg, 1, gsl_rng_uniform(r) * 6 -3);
        gsl_vector_set(r_trg, 2, gsl_rng_uniform(r) * 6 -3);

        gsl_vector_set(a_trg, 0, gsl_rng_uniform(r) * 6 -3);
        gsl_vector_set(a_trg, 1, gsl_rng_uniform(r) * 6 -3);
        gsl_vector_set(a_trg, 2, gsl_rng_uniform(r) * 6 -3);

        gsl_vector_set(r_src, 0, gsl_rng_uniform(r) * 6 -3);
        gsl_vector_set(r_src, 1, gsl_rng_uniform(r) * 6 -3);
        gsl_vector_set(r_src, 2, gsl_rng_uniform(r) * 6 -3);

        gsl_vector_set(a_src, 0, gsl_rng_uniform(r) * 6 -3);
        gsl_vector_set(a_src, 1, gsl_rng_uniform(r) * 6 -3);
        gsl_vector_set(a_src, 2, gsl_rng_uniform(r) * 6 -3);

        //smoothing radius and volume
        double s_trg = gsl_rng_uniform(r);
        double v_trg = gsl_rng_uniform(r);
        double s_src = gsl_rng_uniform(r);
        double v_src = gsl_rng_uniform(r);

        if(!testSingleInteract(nu, s_src, s_trg, r_src, r_trg, a_src, a_trg, v_src, v_trg)){
            std::cout << "Number "<< i << " is wrong! nu:" << nu << " s: " << s_src << ", " << s_trg << " v: " <<v_src << ", " << v_trg << "\n";
            OUT("source pos",r_src);
            OUT("target pos",r_trg);
            OUT("source vor",a_src);
            OUT("target vor",a_trg);
            wrong++;
        }
    }

    std::cout << "Number wrong" << wrong << " (" << (100.0 * wrong) / iterations << " %)";
    gsl_vector_free(r_src);
    gsl_vector_free(a_src);
    gsl_vector_free(r_trg);
    gsl_vector_free(a_trg);
}

void compare_equal(gsl_vector *gpu, gsl_vector *cpu, int size, int offset) {
    int wrong = 0;
    std::cout << "Position\n";
    for(int i = 0; i < size; i++){
        double gpu_v = gsl_vector_get(gpu,i);
        double cpu_v = gsl_vector_get(cpu,i);

        if(0 != gsl_fcmp(gpu_v,cpu_v, epsilon)  && ! ( closeToZero(gpu_v) && closeToZero(cpu_v)) ){
            std::cout << "Different result on gpu (" << gpu_v << ") and cpu (" << cpu_v << ") at index " << i <<".\n";
            wrong++;
        }
    }

    if(offset != 0) {
        std::cout << "Vorticity\n";

        for (int i = offset; i < offset + size; i++) {
            double gpu_v = gsl_vector_get(gpu, i);
            double cpu_v = gsl_vector_get(cpu, i);

            if (0 != gsl_fcmp(gpu_v, cpu_v, epsilon) && ! ( closeToZero(gpu_v) && closeToZero(cpu_v)) ) {
                std::cout << "Different result on gpu (" << gpu_v << ") and cpu (" << cpu_v << ") at index "
                          << i - offset << ".\n";
                wrong++;
            }
        }
    }
    if(wrong == 0){
        std::cout << "No error found";
    } else {
        std::cout << wrong << " differences  (" << (100.0 * wrong) / (size * (offset == 0 ? 1 : 2)) << " %)";
    }
}

bool closeToZero(double gpuV) {
    return abs(gpuV) < 1e-15;
}

void singleStep(){
    unsigned long int seed1 = 53478496;
    unsigned long int seed2 = 3543753850;

    int size1 = 3000;
    int size2 = 4000;


    gsl_rng * r;
    const gsl_rng_type * T;

    gsl_rng_env_setup();
    T = gsl_rng_default;
    r = gsl_rng_alloc (T);

    gsl_rng_set(r, seed1);
    test_wake wakeGPU = test_wake(size1, r);
    gsl_rng_set(r, seed1);
    test_wake wakeCPU = test_wake(size1, r);

    gsl_rng_set(r, seed2);
    test_wake wakeGPU2 = test_wake(size2, r);
    gsl_rng_set(r, seed2);
    test_wake wakeCPU2 = test_wake(size2, r);

    __interaction *interactionGPU = new gpu(&wakeGPU, &wakeGPU2);
    __interaction *interactionCPU = new __parallel(&wakeCPU, &wakeCPU2);

    std::cout << "Solve GPU\n";
    interactionGPU->solve();
    std::cout << "Solve CPU\n";
    interactionCPU->solve();

    std::cout << "compare (with epsilon " << epsilon << ")\n";
    gsl_vector *ratesGPU = gsl_vector_calloc(wakeGPU._size + wakeGPU2._size);
    gsl_vector *ratesCPU = gsl_vector_calloc(wakeCPU._size + wakeCPU2._size);

    interactionGPU->getRates(ratesGPU);
    interactionCPU->getRates(ratesCPU);

    compare_equal(ratesGPU, ratesCPU, wakeGPU._size + wakeGPU2._size, 0);
}

void wholeIntegration(){
    /*
    unsigned long int seed1 = 98;

    int size1 = 3000;

    gsl_rng * r;
    const gsl_rng_type * T;

    gsl_rng_env_setup();
    T = gsl_rng_default;
    r = gsl_rng_alloc (T);

    gsl_rng_set(r, seed1);
    test_wake wakeGPU = test_wake(size1, r);
    gsl_rng_set(r, seed1);
    test_wake wakeCPU = test_wake(size1, r);
*/

    pawan::__io *IOvringGPU = new pawan::__io("testGPU");


    pawan::__wake *W = new pawan::__vring(1.0,0.1,6,117,0.0735);
    pawan::__interaction *S = new pawan::__parallel(W);

    pawan::__resolve *R = new pawan::__resolve();
    S->diagnose();//simply calculate diagnostics
    R->rebuild(S,IOvringGPU);
    W->print();
    S->diagnose();
    S->solve();
    W->print();

    //pawan::__wake *W1 = new pawan::__ring(1.0,5.0,0.1,1000);
    pawan::__wake *wakeGPU = new pawan::__wake(W);
    //pawan::__wake *W2 = new pawan::__ring(1.0,5.0,0.1,1000);
    pawan::__wake *wakeCPU = new pawan::__wake(W);


    pawan::__interaction *SvringGPU = new pawan::__parallel(wakeGPU);
    pawan::__integration *INvringGPU = new pawan::gpu_euler(5,100);

    INvringGPU->integrate(SvringGPU,IOvringGPU,false);

    pawan::__io *IOvringCPU = new pawan::__io("testCPU");

    pawan::__interaction *SvringCPU = new pawan::__parallel(wakeCPU);
    pawan::__integration *INvringCPU = new pawan::__integration(5,100);

    INvringCPU->integrate(SvringCPU,IOvringCPU,false);

    std::cout << "compare (with epsilon " << epsilon << ")\n";
    gsl_vector *statesGPU = gsl_vector_calloc(wakeGPU->_maxsize);
    gsl_vector *statesCPU = gsl_vector_calloc(wakeCPU->_maxsize);

    wakeGPU->getStates(statesGPU);
    wakeCPU->getStates(statesCPU);

    compare_equal(statesGPU, statesCPU, wakeGPU->_numParticles * 3, wakeGPU->_maxsize/2);


}


void test()
{
    //testInteractWithRandomValues();

    //singleStep();

    wholeIntegration();

}


