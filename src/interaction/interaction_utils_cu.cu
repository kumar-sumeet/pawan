#include "hip/hip_runtime.h"
//
// Created by ge56beh on 13.09.21.
//

/*
#ifdef __HIPCC__
#define CUDA_HOSTDEV __host__ __device__
#endif
*/
#include <math.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_vector.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

#include "interaction_utils_cu.h"
#include "test_cuda.h"
#include "la_utils_cu.h"
#include "cuda_utils_cu.h"
#include "src/utils/gsl_utils.h"

#define HIP_PI       3.14159265358979323846	/* pi */
#define CUDART_1_PI     0.31830988618379067154  /* 1/pi */
#define CUDART_SQRT1_2	0.70710678118654752440	/* 1/sqrt(2) */
#define BLOCK_SIZE      64                      /*max value shared memory-limited right now to 64*/

__device__
void cuKERNEL(	const double &rho,
                const double &sigma,
                double &q,
                double &F,
                double &Z){
    double rho_bar = rho/sigma;
    double sig3 = sigma*sigma*sigma;
    double phi = 0.25*CUDART_1_PI*erf(CUDART_SQRT1_2*rho_bar)/sig3;
    Z = 0.5*exp(-0.5*rho_bar*rho_bar)/sig3/pow(HIP_PI,1.5);
    q = (phi/rho_bar - Z)/la_d_pow_2(rho_bar);
    F = (Z - 3*q)/la_d_pow_2(rho);
};

__device__
void cuVELOCITY(  const double &kernel,
                  const double4 *vorticity,
                  const double4 *displacement,
                  double4 *velocity){
    la_d4_cross(vorticity,displacement,velocity);
    la_d4_blas_dscal(kernel,velocity);
};

__device__
void cuVORSTRETCH( const double &q,
                   const double &F,
                   const double4 *source_vorticity,
                   const double4 *target_vorticity,
                   const double4 *displacement,
                   double4 *retvorcity){
    //trgXsrc = a_target x a_source
    double4 trgXsrc = {0.0, 0.0, 0.0, 0.0};
    la_d4_cross(target_vorticity,source_vorticity,&trgXsrc);

    //temp measure, needs fixing!!!
    if (target_vorticity->x == source_vorticity->x  &&  target_vorticity->y == source_vorticity->y  &&
        target_vorticity->z == source_vorticity->z  &&  target_vorticity->w == source_vorticity->w){
        trgXsrc = {0.0,0.0,0.0,0.0,};
    }

    // da/dt = q*(a_trg x a_src)
    double4 crossed = {q*trgXsrc.x, q*trgXsrc.y, q*trgXsrc.z, q*trgXsrc.w};

    // da/dt = F*[disp.(a_trg x a_src)]disp
    double roaxa = 0.0;
    la_d4_blas_ddot(displacement,&trgXsrc,&roaxa);
    double4 stretch = {displacement->x, displacement->y, displacement->z, displacement->w};
    la_d4_blas_dscal(F*roaxa,&stretch);

    la_d4_add(retvorcity,&crossed);
    la_d4_add(retvorcity,&stretch);
};

__device__
void cuDIFFUSION(	 const double nu,
                     const double &sigma,
                     const double &Z,
                     const double4 *source_vorticity,
                     const double4 *target_vorticity,
                     const double &source_volume,
                     const double &target_volume,
                     double4 *retvorcity){
    // va12 = volume_target*vorticity_source
    double4 va12 = {source_vorticity->x, source_vorticity->y, source_vorticity->z, source_vorticity->w};
    la_d4_blas_dscal(target_volume,&va12);

    // va21 = volume_source*vorticity_target
    double4 va21 = {target_vorticity->x, target_vorticity->y, target_vorticity->z, target_vorticity->w};
    la_d4_blas_dscal(source_volume,&va21);

    // dva = 2*nu*Z*(va12 - va21)/sigma^2
    double sig12 = 0.5*sigma*sigma;
    double4 dva = {va12.x, va12.y, va12.z, va12.w};
    la_d4_sub(&dva,&va21);
    la_d4_blas_dscal(Z*nu/sig12,&dva);

    // da = da + dva
    la_d4_add(retvorcity,&dva);
};

__device__
void cuINTERACT( const double nu,
                 const double &s_source,
                 const double &s_target,
                 const double4 *r_source,
                 const double4 *r_target,
                 const double4 *a_source,
                 const double4 *a_target,
                 const double &v_source,
                 const double &v_target,
                 double4 *dr_target,
                 double4 *da_target){
    // Kernel Computation
    double4 displacement = {r_target->x - r_source->x, r_target->y - r_source->y,
                            r_target->z - r_source->z,r_target->w - r_source->w};
    double rho = la_d4_blas_dnrm2_soft(&displacement);
    double q = 0.0, F = 0.0, Z = 0.0;
    double sigma = sqrt(s_source*s_source + s_target*s_target)/2.0;

    // Velocity computation
    double4 *dr = la_d4calloc(1);
    cuKERNEL(rho,sigma,q,F,Z);
    cuVELOCITY(-q,a_target,&displacement,dr);
    la_d4_add(dr_target,dr);

    // Rate of change of vorticity computation
    double4 *da =  la_d4calloc(1);
    cuVORSTRETCH(q,F,a_source,a_target,&displacement,da);
    cuDIFFUSION(nu,sigma,Z,a_source,a_target,v_source,v_target,da);
    la_d4_sub(da_target,da);

    la_d4dealloc(da);
    la_d4dealloc(dr);
}

__device__
void cuinteract(const int threadnum,
                const double nu,
                const double4 *Wpos_d4_arr,
                const double4 *Wvor_d4_arr,
                const double *Wrad_d_vec,
                const double *Wvol_d_vec,
                const int n,
                double4 *rates){

    const double4 r_src = Wpos_d4_arr[threadnum];
    const double4 a_src = Wvor_d4_arr[threadnum];
    const double s_src = Wrad_d_vec[threadnum];
    const double v_src = Wvol_d_vec[threadnum];

    for (int blocknum = 0; blocknum < gridDim.x; blocknum++) {

        __shared__ double4 wpos_d4_block[BLOCK_SIZE], wvor_d4_block[BLOCK_SIZE];
        __shared__ double wrad_d_block[BLOCK_SIZE], wvol_d_block[BLOCK_SIZE];

        wpos_d4_block[threadIdx.x] = Wpos_d4_arr[blocknum * blockDim.x + threadIdx.x];
        wvor_d4_block[threadIdx.x] = Wvor_d4_arr[blocknum * blockDim.x + threadIdx.x];
        wrad_d_block[threadIdx.x] = Wrad_d_vec[blocknum * blockDim.x + threadIdx.x];
        wvol_d_block[threadIdx.x] = Wvol_d_vec[blocknum * blockDim.x + threadIdx.x];
        __syncthreads();
        //#pragma unroll
        for (int j = 0; j < BLOCK_SIZE; j++) {
            idx = {threadnum, blocknum * blockDim.x + threadIdx.x};
            const double4 r_trg = wpos_d4_block[j];
            const double4 a_trg = wvor_d4_block[j];
            const double s_trg = wrad_d_block[j];
            const double v_trg = wvol_d_block[j];

            cuINTERACT(nu, s_src, s_trg, &r_src, &r_trg, &a_src, &a_trg, v_src, v_trg, rates, rates + 1);
        }
        __syncthreads();  //necessary before operations move onto next block
    }
}
/*!
 *  (Cuda kernel) Computes the time-integrated position and vorticity
 */
__global__
void cuINTERACT_rk4(double nu,
                    double4 *Wpos_d4_arr,
                    double4 *Wvor_d4_arr,
                    double4 *Wvel_d4_arr,
                    double4 *Wretvor_d4_arr,
                    double *Wrad_d_vec,
                    double *Wvol_d_vec,
                    double4 *Wpos_d4_arr_x1,
                    double4 *Wvor_d4_arr_x1,
                    double4 *Wpos_d4_arr_x2,
                    double4 *Wvor_d4_arr_x2,
                    double4 *Wpos_d4_arr_x3,
                    double4 *Wvor_d4_arr_x3,
                    const int n,
                    const double dt,
                    const size_t t_steps){
    cg::grid_group grid = cg::this_grid();
    //cg::thread_block block = cg::this_thread_block();

    int threadnum = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadnum < n) {
        for (size_t t_n = 1; t_n<=t_steps; ++t_n) {

            double4 *k1 = la_d4calloc(2);
            double4 *k2 = la_d4calloc(2);
            double4 *k3 = la_d4calloc(2);
            double4 *k4 = la_d4calloc(2);

//######################################################################################################################
            cuinteract(threadnum, nu, Wpos_d4_arr, Wvor_d4_arr, Wrad_d_vec, Wvol_d_vec, n, k1);

            cg::sync(grid);
            la_d4_set(threadnum, Wpos_d4_arr_x1, k1);    la_d4_set(threadnum, Wvor_d4_arr_x1, k1+1);
            la_d4_blas_dscal(threadnum,0.5*dt, Wpos_d4_arr_x1);   la_d4_blas_dscal(threadnum,0.5*dt, Wvor_d4_arr_x1);
            la_d4_add(threadnum, Wpos_d4_arr_x1, Wpos_d4_arr+threadnum);    la_d4_add(threadnum, Wvor_d4_arr_x1, Wvor_d4_arr+threadnum);

            cg::sync(grid);
//######################################################################################################################
            cuinteract(threadnum, nu, Wpos_d4_arr_x1, Wvor_d4_arr_x1, Wrad_d_vec, Wvol_d_vec, n, k2);

            cg::sync(grid);
            la_d4_set(threadnum, Wpos_d4_arr_x2, k2);    la_d4_set(threadnum, Wvor_d4_arr_x2, k2+1);
            la_d4_blas_dscal(threadnum,0.5*dt, Wpos_d4_arr_x2);   la_d4_blas_dscal(threadnum,0.5*dt, Wvor_d4_arr_x2);
            la_d4_add(threadnum, Wpos_d4_arr_x2, Wpos_d4_arr+threadnum);    la_d4_add(threadnum, Wvor_d4_arr_x2, Wvor_d4_arr+threadnum);

            cg::sync(grid);
//######################################################################################################################
            cuinteract(threadnum, nu, Wpos_d4_arr_x2, Wvor_d4_arr_x2, Wrad_d_vec, Wvol_d_vec, n, k3);

            cg::sync(grid);
            la_d4_set(threadnum, Wpos_d4_arr_x3, k3);    la_d4_set(threadnum, Wvor_d4_arr_x3, k3+1);
            la_d4_blas_dscal(threadnum,dt, Wpos_d4_arr_x3);   la_d4_blas_dscal(threadnum,dt, Wvor_d4_arr_x3);
            la_d4_add(threadnum, Wpos_d4_arr_x3, Wpos_d4_arr+threadnum);    la_d4_add(threadnum, Wvor_d4_arr_x3, Wvor_d4_arr+threadnum);

            cg::sync(grid);
//######################################################################################################################
            cuinteract(threadnum, nu, Wpos_d4_arr_x3, Wvor_d4_arr_x3, Wrad_d_vec, Wvol_d_vec, n, k4);

            cg::sync(grid);
            la_d4_add(k1,k4,2);   la_d4_blas_dscal(dt/6.0,k1,2);
            la_d4_add(k2,k3,2);   la_d4_blas_dscal(dt/3.0,k2,2);
            la_d4_add(k1,k2,2);
            la_d4_add(threadnum, Wpos_d4_arr, k1);    la_d4_add(threadnum, Wvor_d4_arr, k1+1);
            la_d4_set(threadnum, Wvel_d4_arr, k4);    la_d4_set(threadnum, Wretvor_d4_arr, k4 + 1);

            cg::sync(grid);
//######################################################################################################################

            la_d4dealloc(k1); la_d4dealloc(k2); la_d4dealloc(k3); la_d4dealloc(k4);

            if (threadnum == 1){ printf("Step %d \n", t_n);}
        }
   }
};

void cuda_main(double &nu,
               double **Wpos_arr,
               double **Wvor_arr,
               double **Wvel_arr,
               double **Wretvor_arr,
               double *Wrad_vec,
               double *Wvol_vec,
               const size_t numParticles,
               const size_t numDimensions,
               const double dt,
               const size_t t_steps){

    int num_gpu = 0;  // number of CUDA GPUs
    hipGetDeviceCount(&num_gpu); //get number of gpus available
    if (num_gpu < 1) {
        printf("no CUDA capable GPUs were detected \n");
        return;
    } else {
        printf("%d CUDA capable GPUs were detected \n", num_gpu);
    }
    int gpuID = num_gpu - 1; //the last (non-default) gpu is 'usually' free
    if (hipSetDevice(gpuID) != hipSuccess)
        printf("something went wrong setting gpu num %d \n", gpuID);
    hipDeviceProp_t cudprop;
    cudaDeviceProperties_print(&cudprop, gpuID);
    printf(" \n \n ");

    int d4_bytes = numParticles * sizeof(double4);    //double4 type heap allocation on device
    int d_bytes = numParticles * sizeof(double);     //double type heap allocation on device

    double4 *Wpos_d4_arr = la_to_d4alloc(Wpos_arr, numParticles, numDimensions);
    double4 *dev_wpos;
    hipMalloc(&dev_wpos, d4_bytes);
    hipMemcpy(dev_wpos, Wpos_d4_arr, d4_bytes, hipMemcpyHostToDevice);

    double4 *Wvor_d4_arr = la_to_d4alloc(Wvor_arr, numParticles, numDimensions);
    double4 *dev_wvor;
    hipMalloc(&dev_wvor, d4_bytes);
    hipMemcpy(dev_wvor, Wvor_d4_arr, d4_bytes, hipMemcpyHostToDevice);

    double4 *Wvel_d4_arr = la_to_d4alloc(Wvel_arr, numParticles, numDimensions);
    double4 *dev_wvel;
    hipMalloc(&dev_wvel, d4_bytes);
    hipMemcpy(dev_wvel, Wvel_d4_arr, d4_bytes, hipMemcpyHostToDevice);

    double4 *Wretvor_d4_arr = la_to_d4alloc(Wretvor_arr, numParticles, numDimensions);
    double4 *dev_wretvor;
    hipMalloc(&dev_wretvor, d4_bytes);
    hipMemcpy(dev_wretvor, Wretvor_d4_arr, d4_bytes, hipMemcpyHostToDevice);

    double4 *dev_wpos_x1, *dev_wpos_x2, *dev_wpos_x3, *dev_wpos_x4;
    hipMalloc(&dev_wpos_x1, d4_bytes); hipMalloc(&dev_wpos_x2, d4_bytes); hipMalloc(&dev_wpos_x3, d4_bytes); hipMalloc(&dev_wpos_x4, d4_bytes);
    hipMemset(dev_wpos_x1,0.0,d4_bytes); hipMemset(dev_wpos_x2,0.0,d4_bytes); hipMemset(dev_wpos_x3,0.0,d4_bytes); hipMemset(dev_wpos_x4,0.0,d4_bytes);

    double4 *dev_wvor_x1, *dev_wvor_x2, *dev_wvor_x3, *dev_wvor_x4;
    hipMalloc(&dev_wvor_x1, d4_bytes); hipMalloc(&dev_wvor_x2, d4_bytes); hipMalloc(&dev_wvor_x3, d4_bytes); hipMalloc(&dev_wvor_x4, d4_bytes);
    hipMemset(dev_wvor_x1,0.0,d4_bytes); hipMemset(dev_wvor_x2,0.0,d4_bytes); hipMemset(dev_wvor_x3,0.0,d4_bytes); hipMemset(dev_wvor_x4,0.0,d4_bytes);

    double *dev_wrad;
    hipMalloc(&dev_wrad, d_bytes);
    hipMemcpy(dev_wrad, Wrad_vec, d_bytes, hipMemcpyHostToDevice);

    double *dev_wvol;
    hipMalloc(&dev_wvol, d_bytes);
    hipMemcpy(dev_wvol, Wvol_vec, d_bytes, hipMemcpyHostToDevice);

    int nBlocks = (numParticles + BLOCK_SIZE - 1) / BLOCK_SIZE;

    //cuda kernel arguments
    void *kernelArgs[] = {
            (void *)&nu,  (void *)&dev_wpos, (void *)&dev_wvor, (void *)&dev_wvel,
            (void *)&dev_wretvor, (void *)&dev_wrad, (void *)&dev_wvol,
            (void *)&dev_wpos_x1, (void *)&dev_wvor_x1,
            (void *)&dev_wpos_x2, (void *)&dev_wvor_x2,  (void *)&dev_wpos_x3, (void *)&dev_wvor_x3,
            (void *)&numParticles, (void *)&dt,  (void *)&t_steps
    };
    dim3 dimGrid(nBlocks,1,1);  //check cuda-samples (multiple blocks possible per SM)
    dim3 dimBlock(BLOCK_SIZE,1,1);

    //lauching cuda kernels with 'cooperative groups'
    hipLaunchCooperativeKernel((void *) cuINTERACT_rk4, dimGrid, dimBlock, kernelArgs);

    //hipDeviceSynchronize();
    hipMemcpy(Wpos_d4_arr, dev_wpos, d4_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(Wvor_d4_arr, dev_wvor, d4_bytes, hipMemcpyDeviceToHost);
    //hipMemcpy(Wrad_vec, dev_wrad, d4_bytes, hipMemcpyDeviceToHost);
    //hipMemcpy(Wvol_vec, dev_wvol, d4_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(Wvel_d4_arr, dev_wvel, d4_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(Wretvor_d4_arr, dev_wretvor, d4_bytes, hipMemcpyDeviceToHost);

    printf("After CUDA---------------------- \n");
    printf("Wpos_arr-----------------\n");
    la_d4print(Wpos_d4_arr, numParticles);
    printf("Wvor_arr-----------------\n");
    la_d4print(Wvor_d4_arr, numParticles);
    printf("Wvel_arr-----------------\n");
    la_d4print(Wvel_d4_arr, numParticles);
    printf("Wretvor_arr-----------------\n");
    la_d4print(Wretvor_d4_arr, numParticles);

    la_to_d4alloc_reverse(Wpos_arr,Wpos_d4_arr ,numParticles,numDimensions);
    la_to_d4alloc_reverse(Wvor_arr,Wvor_d4_arr ,numParticles,numDimensions);
    la_to_d4alloc_reverse(Wvel_arr,Wvel_d4_arr ,numParticles,numDimensions);
    la_to_d4alloc_reverse(Wretvor_arr,Wretvor_d4_arr ,numParticles,numDimensions);

    free(Wpos_d4_arr);
    free(Wvor_d4_arr);
    free(Wvel_d4_arr);
    free(Wretvor_d4_arr);

    hipFree(dev_wpos);
    hipFree(dev_wvor);
    hipFree(dev_wvel);
    hipFree(dev_wretvor);
    hipFree(dev_wrad);
    hipFree(dev_wvol);
}

