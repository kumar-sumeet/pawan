#include "hip/hip_runtime.h"
#include "testcuda1.h"

typedef struct { float4 *pos, *vel; } BodySystem;

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__
void bodyForce(float4 *p, float4 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

        for (int tile = 0; tile < gridDim.x; tile++) {
            __shared__ float3 spos[BLOCK_SIZE];
            float4 tpos = p[tile * blockDim.x + threadIdx.x];
            spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
            __syncthreads();

#pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j++) {
                float dx = spos[j].x - p[i].x;
                float dy = spos[j].y - p[i].y;
                float dz = spos[j].z - p[i].z;
                float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                float invDist = rsqrtf(distSqr);
                float invDist3 = invDist * invDist * invDist;

                Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
            }
            __syncthreads();
        }

        v[i].x += dt*Fx; v[i].y += dt*Fy; v[i].z += dt*Fz;
    }
}

void cudaDeviceProperties_print(hipDeviceProp_t *cudprop, const int &gpuID){
    hipGetDeviceProperties(cudprop, gpuID);
    printf("-----------device properties------------- \n");
    printf("| \t hipDeviceProp_t.name                : \t %s \n", cudprop->name);
    printf("| \t hipDeviceProp_t.maxThreadsPerBlock  : \t %d \n", cudprop->maxThreadsPerBlock);
    printf("| hipDeviceProp_t.maxThreadsPerMultiProcessor: \t %d \n", cudprop->maxThreadsPerMultiProcessor);
    // printf("| hipDeviceProp_t.maxBlocksPerMultiProcessor : \t %d \n", cudprop->maxBlocksPerMultiProcessor);
    printf("| \t hipDeviceProp_t.maxThreadsDim       : \t %d,%d,%d \n", cudprop->maxThreadsDim[0],
           cudprop->maxThreadsDim[1],
           cudprop->maxThreadsDim[2]);
    printf("| \t hipDeviceProp_t.maxGridSize         : \t %d,%d,%d \n", cudprop->maxGridSize[0],
           cudprop->maxGridSize[1],
           cudprop->maxGridSize[2]);
    printf("| \t hipDeviceProp_t.warpSize            : \t %d \n", cudprop->warpSize);
    printf("----------------------------------------- \n");

}

void testcuda_call(){

    printf("---------------------Entering testcuda_call()----------------------------");
    int num_gpu = 0;  // number of CUDA GPUs
    printf("Launching CUDA computation... \n\n");
    hipGetDeviceCount(&num_gpu); //get number of gpus available
    if (num_gpu < 1) {
        printf("no CUDA capable GPUs were detected \n");
        return ;
    } else {
        printf("%d CUDA capable GPUs were detected \n", num_gpu);
    }
    int gpuID = num_gpu-1; //the last (non-default) gpu is 'usually' free
    if (hipSetDevice(gpuID) != hipSuccess)
        printf("something went wrong setting gpu num %d \n", gpuID);
    hipDeviceProp_t cudprop;
    //cudaDeviceProperties_print(&cudprop, gpuID);


    printf("\n \n ");

    int nIters = 1;
    int nBodies;

    for (int iter = 1; iter <= nIters; iter++) {
        nBodies = pow(2,20);
        const float dt = 0.01f; // time step
        printf("nBodies                   = %d \n", nBodies);

        int bytes = 2*nBodies*sizeof(float4);
        std::cout << bytes << std::endl;
        float *buf = (float*)malloc(bytes);
        BodySystem p = { (float4*)buf, ((float4*)buf) + nBodies };

        randomizeBodies(buf, 8*nBodies); // Init pos / vel data

        float *d_buf;
        hipMalloc(&d_buf, bytes);
        BodySystem d_p = { (float4*)d_buf, ((float4*)d_buf) + nBodies };

        int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
        printf("No. of BLOCKS               = %d \n No. of THREADS per BLOCK = %d \n", nBlocks, BLOCK_SIZE);

        hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
        bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p.pos, d_p.vel, dt, nBodies);
        hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
        printf("CUDA compilation finished... \n");
        for (int i = 0 ; i < nBodies; i++) { // integrate position
            p.pos[i].x += p.vel[i].x*dt;
            p.pos[i].y += p.vel[i].y*dt;
            p.pos[i].z += p.vel[i].z*dt;
        }

        free(buf);
        hipFree(d_buf);
        printf("--------------------- \n");
    }
}
