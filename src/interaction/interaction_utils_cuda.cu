#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "src/integration/integration.h"
#include "src/io/io.h"
#include "src/utils/timing_utils.h"
#include "src/wake/wake_struct.h"

#define BLOCKSIZE 256
#define SOFTENING_CUDA 1e-12f
#define FACTOR1 0.5
#define FACTOR2 1.0

__device__ void KERNEL_CUDA(const double& rho,
                            const double& sigma,
                            double& q,
                            double& F,
                            double& Z) {
    double rho_bar = rho / sigma;
    double sig3 = sigma * sigma * sigma;
    double phi = 0.25 * M_1_PI * erf(M_SQRT1_2 * rho_bar) / sig3;
    Z = 0.5 * exp(-0.5 * rho_bar * rho_bar) / sig3 / pow(M_PI, 1.5);
    q = (phi / rho_bar - Z) / (rho_bar * rho_bar);
    F = (Z - 3 * q) / (rho * rho);
};

__device__ void VELOCITY_CUDA(const double& kernel,
                              const double* vorticity,
                              const float3 displacement,
                              float3& velocity) {
    velocity.x = (vorticity[1] * displacement.z - vorticity[2] * displacement.y) * kernel;
    velocity.y = (vorticity[2] * displacement.x - vorticity[0] * displacement.z) * kernel;
    velocity.z = (vorticity[0] * displacement.y - vorticity[1] * displacement.x) * kernel;
};

__device__ void VORSTRETCH_CUDA(const double& q,
                                const double& F,
                                const double* source_vorticity,
                                const double* target_vorticity,
                                const float3 displacement,
                                float3& retvorcity) {
    float3 trgXsrc = make_float3(target_vorticity[1] * source_vorticity[2] - target_vorticity[2] * source_vorticity[1],
                                 target_vorticity[1] * source_vorticity[2] - target_vorticity[2] * source_vorticity[1],
                                 target_vorticity[0] * source_vorticity[1] - target_vorticity[1] * source_vorticity[0]);
    double roaxa = 0.0;
    roaxa += displacement.x * trgXsrc.x;
    roaxa += displacement.y * trgXsrc.y;
    roaxa += displacement.z * trgXsrc.z;

    retvorcity.x += ((trgXsrc.x * q) + (displacement.x * F * roaxa));
    retvorcity.y += ((trgXsrc.y * q) + (displacement.y * F * roaxa));
    retvorcity.z += ((trgXsrc.z * q) + (displacement.z * F * roaxa));
};

__device__ void DIFFUSION_CUDA(const double& nu,
                               const double& sigma,
                               const double& Z,
                               const double* source_vorticity,
                               const double* target_vorticity,
                               const double& source_volume,
                               const double& target_volume,
                               float3& retvorcity) {
    double sig12 = 0.5 * sigma * sigma;
    retvorcity.x += ((source_vorticity[0] * target_volume) - (target_vorticity[0] * source_volume)) * (Z * nu / sig12);
    retvorcity.y += ((source_vorticity[1] * target_volume) - (target_vorticity[1] * source_volume)) * (Z * nu / sig12);
    retvorcity.z += ((source_vorticity[2] * target_volume) - (target_vorticity[2] * source_volume)) * (Z * nu / sig12);
}

__device__ void INTERACT_CUDA(
    const double& nu,
    const double& s_source,
    const double& s_target,
    const double* r_source,
    const double* r_target,
    const double* a_source,
    const double* a_target,
    const double& v_source,
    const double& v_target,
    double* dr_source,
    double* da_source) {
    // kenerl computation
    float3 displacement = make_float3(r_target[0] - r_source[0], r_target[1] - r_source[1], r_target[2] - r_source[2]);
    double rho = std::sqrt(displacement.x * displacement.x + displacement.y * displacement.y + displacement.z * displacement.z + SOFTENING);
    double q = 0.0, F = 0.0, Z = 0.0;
    double sigma = std::sqrt(s_source * s_source + s_target * s_target) / 2.0;

    // velocity computation
    float3 dr = make_float3(0.0, 0.0, 0.0);
    KERNEL_CUDA(rho, sigma, q, F, Z);
    VELOCITY_CUDA(q, a_source, displacement, dr);
    dr_source[0] += dr.x;
    dr_source[1] += dr.y;
    dr_source[2] += dr.z;

    // Rate of change of vorticity computation
    float3 da = make_float3(0.0, 0.0, 0.0);
    VORSTRETCH_CUDA(q, F, a_source, a_target, displacement, da);
    DIFFUSION_CUDA(nu, sigma, Z, a_source, a_target, v_source, v_target, da);
    da_source[0] -= da.x;
    da_source[1] -= da.y;
    da_source[2] -= da.z;
}

__global__ void setStates_cuda(pawan::wake_cuda w, const double* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < w.size / 2) {
        w.position[tid] = state[tid];
        w.vorticity[tid] = state[tid + w.size / 2];
    }
}

__global__ void getStates_cuda(pawan::wake_cuda w, double* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < w.size / 2) {
        state[tid] = w.position[tid];
        state[tid + w.size / 2] = w.vorticity[tid];
    }
}

__global__ void getRates_cuda(pawan::wake_cuda w, double* rate) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < w.size / 2) {
        rate[tid] = w.velocity[tid];
        rate[tid + w.size / 2] = w.retvorcity[tid];
    }
}

__global__ void clear(pawan::wake_cuda w) {
    size_t numDimensions = w.numDimensions;
    size_t numParticles = w.numParticles;
    for (size_t i = 0; i < numParticles; i++)
        for (size_t j = 0; j < numDimensions; j++)
            w.velocity[i * numDimensions + j] = w.retvorcity[i * numDimensions + j] = 0.0;
}

__global__ void interact_cuda(pawan::wake_cuda w) {
    int i_src = blockIdx.x * blockDim.x + threadIdx.x;
    size_t numDimensions = w.numDimensions;

    double *r_src, *a_src, *dr_src, *da_src;
    double s_src, v_src;
    if (i_src < w.numParticles) {
        r_src = &(w.position[i_src * numDimensions]);
        a_src = &(w.vorticity[i_src * numDimensions]);
        dr_src = &(w.velocity[i_src * numDimensions]);
        da_src = &(w.retvorcity[i_src * numDimensions]);
        s_src = w.radius[i_src];
        v_src = w.volume[i_src];
    }

    __shared__ double r_trgs[BLOCKSIZE * 3];
    __shared__ double a_trgs[BLOCKSIZE * 3];
    __shared__ double s_trgs[BLOCKSIZE];
    __shared__ double v_trgs[BLOCKSIZE];

    // Loop over tiles
    for (int tile = 0; tile < (w.numParticles + BLOCKSIZE - 1) / BLOCKSIZE; tile++) {
        int index = threadIdx.x + tile * BLOCKSIZE;

        // Load data from global memory to shared memory
        if (index < w.numParticles) {
            r_trgs[threadIdx.x * numDimensions] = w.position[index * numDimensions];
            r_trgs[threadIdx.x * numDimensions + 1] = w.position[index * numDimensions + 1];
            r_trgs[threadIdx.x * numDimensions + 2] = w.position[index * numDimensions + 2];

            a_trgs[threadIdx.x * numDimensions] = w.vorticity[index * numDimensions];
            a_trgs[threadIdx.x * numDimensions + 1] = w.vorticity[index * numDimensions + 1];
            a_trgs[threadIdx.x * numDimensions + 2] = w.vorticity[index * numDimensions + 2];

            s_trgs[threadIdx.x] = w.radius[index];
            v_trgs[threadIdx.x] = w.volume[index];
        }

        __syncthreads();

        // Compute interactions within the block
        int num_targets = (BLOCKSIZE < w.numParticles - tile * BLOCKSIZE) ? BLOCKSIZE : w.numParticles - tile * BLOCKSIZE;
        if (i_src < w.numParticles) {
            for (size_t i_trg = 0; i_trg < num_targets; i_trg++) {
                const double* r_trg = &(r_trgs[i_trg * numDimensions]);
                const double* a_trg = &(a_trgs[i_trg * numDimensions]);
                double s_trg = s_trgs[i_trg];
                double v_trg = v_trgs[i_trg];

                INTERACT_CUDA(w._nu, s_src, s_trg, r_src, r_trg, a_src, a_trg, v_src, v_trg, dr_src, da_src);
            }
        }

        __syncthreads();
    }
}

__global__ void rk4_process(const double dt, double* x, const double* k, const double* d_states, const double factor, const int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < len) {
        x[tid] = k[tid];
        x[tid] *= factor * dt;
        x[tid] += d_states[tid];
    }
}

__global__ void rk4_final(const double dt, double* d_states, double* k1, double* k2, const double* k3, const double* k4, const int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < len) {
        k1[tid] += k4[tid];
        k1[tid] *= dt / 6.;

        k2[tid] += k3[tid];
        k2[tid] *= dt / 3.;

        k1[tid] += k2[tid];

        d_states[tid] += k1[tid];
    }
}

void step_cuda(const double dt, pawan::wake_cuda* w, double* d_states, double* x1, double* x2, double* x3, double* k1, double* k2, double* k3, double* k4, const int len) {
    hipMemcpy(x1, d_states, sizeof(double) * len, hipMemcpyDeviceToDevice);

    int numBlocks_states = (len / 2 + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlocks_rk = (len + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlocks_interact = (w->numParticles + BLOCKSIZE - 1) / BLOCKSIZE;

    // k1 = f(x,t)
    setStates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, d_states);
    clear<<<1, 1>>>(*w);
    interact_cuda<<<numBlocks_interact, BLOCKSIZE>>>(*w);
    getRates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, k1);

    // x1 = x + 0.5*dt*k1
    rk4_process<<<numBlocks_rk, BLOCKSIZE>>>(dt, x1, k1, d_states, FACTOR1, len);

    // k2 = f(x1, t+0.5*dt)
    setStates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, x1);
    clear<<<1, 1>>>(*w);
    interact_cuda<<<numBlocks_interact, BLOCKSIZE>>>(*w);
    getRates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, k2);

    // x2 = x1 + 0.5*dt*dx2
    rk4_process<<<numBlocks_rk, BLOCKSIZE>>>(dt, x2, k2, d_states, FACTOR1, len);

    // k3 = f(x2, t+0.5*dt)
    setStates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, x2);
    clear<<<1, 1>>>(*w);
    interact_cuda<<<numBlocks_interact, BLOCKSIZE>>>(*w);
    getRates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, k3);

    // x3 = x2 + dt*k3
    rk4_process<<<numBlocks_rk, BLOCKSIZE>>>(dt, x3, k3, d_states, FACTOR2, len);

    // k4 = f(x3, t+dt)
    setStates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, x3);
    clear<<<1, 1>>>(*w);
    interact_cuda<<<numBlocks_interact, BLOCKSIZE>>>(*w);
    getRates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, k4);

    rk4_final<<<numBlocks_rk, BLOCKSIZE>>>(dt, d_states, k1, k2, k3, k4, len);

    setStates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, d_states);
}

extern "C" void cuda_step_wrapper(const double _dt, pawan::wake_struct* w, double* state_array) {
    pawan::wake_cuda cuda_wake;
    cuda_wake.size = w->size;
    cuda_wake.numParticles = w->numParticles;
    hipMallocManaged(&cuda_wake.position, sizeof(double) * w->size);
    hipMallocManaged(&cuda_wake.velocity, sizeof(double) * w->size);
    hipMallocManaged(&cuda_wake.vorticity, sizeof(double) * w->size);
    hipMallocManaged(&cuda_wake.retvorcity, sizeof(double) * w->size);
    hipMallocManaged(&cuda_wake.radius, sizeof(double) * w->numParticles);
    hipMallocManaged(&cuda_wake.volume, sizeof(double) * w->numParticles);
    hipMallocManaged(&cuda_wake.birthstrength, sizeof(double) * w->numParticles);

    // initialize wake on gpu
    for (size_t i = 0; i < w->numParticles; i++) {
        cuda_wake.radius[i] = w->radius[i];
        cuda_wake.volume[i] = w->volume[i];
        cuda_wake.birthstrength[i] = w->birthstrength[i];
        for (size_t j = 0; j < w->numDimensions; j++) {
            cuda_wake.position[i * w->numDimensions + j] = w->position[i][j];
            cuda_wake.velocity[i * w->numDimensions + j] = w->velocity[i][j];
            cuda_wake.vorticity[i * w->numDimensions + j] = w->vorticity[i][j];
            cuda_wake.retvorcity[i * w->numDimensions + j] = w->retvorcity[i][j];
        }
    }

    // states, ks and xs
    double* d_states;
    hipMalloc(&d_states, sizeof(double) * w->size);
    hipMemcpy(d_states, state_array, sizeof(double) * w->size, hipMemcpyHostToDevice);

    double *x1, *x2, *x3, *k1, *k2, *k3, *k4;
    hipMalloc(&x1, sizeof(double) * w->size);
    hipMalloc(&x2, sizeof(double) * w->size);
    hipMalloc(&x3, sizeof(double) * w->size);
    hipMalloc(&k1, sizeof(double) * w->size);
    hipMalloc(&k2, sizeof(double) * w->size);
    hipMalloc(&k3, sizeof(double) * w->size);
    hipMalloc(&k4, sizeof(double) * w->size);

    double tStart = TIME();
    for (size_t i = 1; i <= STEPS; i++) {
        OUT("\tStep", i);
        step_cuda(_dt, &cuda_wake, d_states, x1, x2, x3, k1, k2, k3, k4, w->size);
    }
    hipDeviceSynchronize();
    double tEnd = TIME();
    OUT("Total Time (s)", tEnd - tStart);

    for (size_t i = 0; i < w->numParticles; i++) {
        std::cout << cuda_wake.position[i * 3] << " " << cuda_wake.position[i * 3 + 1] << " " << cuda_wake.position[i * 3 + 2] << " " << std::endl;
    }

    hipFree(d_states);
    hipFree(x1);
    hipFree(x2);
    hipFree(x3);
    hipFree(k1);
    hipFree(k2);
    hipFree(k3);
    hipFree(k4);
    hipFree(cuda_wake.radius);
    hipFree(cuda_wake.volume);
    hipFree(cuda_wake.birthstrength);
    hipFree(cuda_wake.position);
    hipFree(cuda_wake.velocity);
    hipFree(cuda_wake.vorticity);
    hipFree(cuda_wake.retvorcity);
}