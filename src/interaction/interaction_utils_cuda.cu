#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "src/integration/integration.h"
#include "src/io/io.h"
#include "src/utils/timing_utils.h"
#include "src/wake/wake_struct.h"

#define BLOCKSIZE 512
#define SOFTENING_CUDA 1e-12f
#define FACTOR1 0.5
#define FACTOR2 1.0

__device__ void KERNEL_CUDA(const double& rho,
                            const double& sigma,
                            double& q,
                            double& F,
                            double& Z) {
    double rho_bar = rho / sigma;
    double sig3 = sigma * sigma * sigma;
    double phi = 0.25 * M_1_PI * erf(M_SQRT1_2 * rho_bar) / sig3;
    Z = 0.5 * exp(-0.5 * rho_bar * rho_bar) / sig3 / pow(M_PI, 1.5);
    q = (phi / rho_bar - Z) / (rho_bar * rho_bar);
    F = (Z - 3 * q) / (rho * rho);
};

__device__ void VELOCITY_CUDA(const double& kernel,
                              const double* vorticity,
                              const float3 displacement,
                              float3& velocity) {
    velocity.x = (vorticity[1] * displacement.z - vorticity[2] * displacement.y) * kernel;
    velocity.y = (vorticity[2] * displacement.x - vorticity[0] * displacement.z) * kernel;
    velocity.z = (vorticity[0] * displacement.y - vorticity[1] * displacement.x) * kernel;
};

__device__ void VORSTRETCH_CUDA(const double& q,
                                const double& F,
                                const double* source_vorticity,
                                const double* target_vorticity,
                                const float3 displacement,
                                float3& retvorcity) {
    double trgXsrc0, trgXsrc1, trgXsrc2;
    trgXsrc0 = target_vorticity[1] * source_vorticity[2] - target_vorticity[2] * source_vorticity[1];
    trgXsrc1 = target_vorticity[2] * source_vorticity[0] - target_vorticity[0] * source_vorticity[2];
    trgXsrc2 = target_vorticity[0] * source_vorticity[1] - target_vorticity[1] * source_vorticity[0];

    double roaxa = 0.0;
    roaxa += displacement.x * trgXsrc0;
    roaxa += displacement.y * trgXsrc1;
    roaxa += displacement.z * trgXsrc2;

    retvorcity.x += ((trgXsrc0 * q) + (displacement.x * F * roaxa));
    retvorcity.y += ((trgXsrc1 * q) + (displacement.y * F * roaxa));
    retvorcity.z += ((trgXsrc2 * q) + (displacement.z * F * roaxa));
};

__device__ void DIFFUSION_CUDA(const double& nu,
                               const double& sigma,
                               const double& Z,
                               const double* source_vorticity,
                               const double* target_vorticity,
                               const double& source_volume,
                               const double& target_volume,
                               float3& retvorcity) {
    double sig12 = 0.5 * sigma * sigma;
    retvorcity.x += ((source_vorticity[0] * target_volume) - (target_vorticity[0] * source_volume)) * (Z * nu / sig12);
    retvorcity.y += ((source_vorticity[1] * target_volume) - (target_vorticity[1] * source_volume)) * (Z * nu / sig12);
    retvorcity.z += ((source_vorticity[2] * target_volume) - (target_vorticity[2] * source_volume)) * (Z * nu / sig12);
}

__device__ void INTERACT_CUDA(
    const double& nu,
    const double& s_source,
    const double& s_target,
    const double* r_source,
    const double* r_target,
    const double* a_source,
    const double* a_target,
    const double& v_source,
    const double& v_target,
    double* dr_source,
    double* da_source) {
    // kenerl computation
    float3 displacement = make_float3(r_target[0] - r_source[0], r_target[1] - r_source[1], r_target[2] - r_source[2]);
    double rho = std::sqrt(displacement.x * displacement.x + displacement.y * displacement.y + displacement.z * displacement.z + SOFTENING);
    double q = 0.0, F = 0.0, Z = 0.0;
    double sigma = std::sqrt(s_source * s_source + s_target * s_target) / 2.0;

    // velocity computation
    float3 dr = make_float3(0.0, 0.0, 0.0);
    KERNEL_CUDA(rho, sigma, q, F, Z);
    VELOCITY_CUDA(q, a_source, displacement, dr);
    dr_source[0] += dr.x;
    dr_source[1] += dr.y;
    dr_source[2] += dr.z;

    // Rate of change of vorticity computation
    float3 da = make_float3(0.0, 0.0, 0.0);
    VORSTRETCH_CUDA(q, F, a_source, a_target, displacement, da);
    DIFFUSION_CUDA(nu, sigma, Z, a_source, a_target, v_source, v_target, da);
    da_source[0] -= da.x;
    da_source[1] -= da.y;
    da_source[2] -= da.z;
}

__global__ void setStates_cuda(pawan::wake_cuda w, const double* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < w.size / 2) {
        w.position[tid] = state[tid];
        w.vorticity[tid] = state[tid + w.size / 2];
    }
}

__global__ void getStates_cuda(pawan::wake_cuda w, double* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < w.size / 2) {
        state[tid] = w.position[tid];
        state[tid + w.size / 2] = w.vorticity[tid];
    }
}

__global__ void getRates_cuda(pawan::wake_cuda w, double* rate) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < w.size / 2) {
        rate[tid] = w.velocity[tid];
        rate[tid + w.size / 2] = w.retvorcity[tid];
    }
}

__global__ void clear(pawan::wake_cuda w) {
    size_t numDimensions = w.numDimensions;
    size_t numParticles = w.numParticles;
    for (size_t i = 0; i < numParticles; i++)
        for (size_t j = 0; j < numDimensions; j++)
            w.velocity[i * numDimensions + j] = w.retvorcity[i * numDimensions + j] = 0.0;
}

__global__ void interact_cuda(pawan::wake_cuda w) {
    int i_src = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_src < w.numParticles) {
        size_t numDimensions = w.numDimensions;

        const double* r_src = &(w.position[i_src * numDimensions]);
        const double* a_src = &(w.vorticity[i_src * numDimensions]);
        double* dr_src = &(w.velocity[i_src * numDimensions]);
        double* da_src = &(w.retvorcity[i_src * numDimensions]);
        double s_src = w.radius[i_src];
        double v_src = w.volume[i_src];

        __shared__ double r_trgs[BLOCKSIZE * 3];
        __shared__ double a_trgs[BLOCKSIZE * 3];
        __shared__ double s_trgs[BLOCKSIZE];
        __shared__ double v_trgs[BLOCKSIZE];
        for (int block = 0; block < gridDim.x; block++) {
            int index = threadIdx.x + block * BLOCKSIZE;
            r_trgs[threadIdx.x * numDimensions] = w.position[index * numDimensions];
            r_trgs[threadIdx.x * numDimensions + 1] = w.position[index * numDimensions + 1];
            r_trgs[threadIdx.x * numDimensions + 2] = w.position[index * numDimensions + 2];

            a_trgs[threadIdx.x * numDimensions] = w.vorticity[index * numDimensions];
            a_trgs[threadIdx.x * numDimensions + 1] = w.vorticity[index * numDimensions + 1];
            a_trgs[threadIdx.x * numDimensions + 2] = w.vorticity[index * numDimensions + 2];

            s_trgs[threadIdx.x] = w.radius[index];
            v_trgs[threadIdx.x] = w.volume[index];
            __syncthreads();
#pragma unroll
            for (size_t i_trg = 0; i_trg < w.numParticles; i_trg++) {
                const double* r_trg = &(r_trgs[i_trg * numDimensions]);
                const double* a_trg = &(a_trgs[i_trg * numDimensions]);
                double s_trg = s_trgs[i_trg];
                double v_trg = v_trgs[i_trg];

                INTERACT_CUDA(w._nu, s_src, s_trg, r_src, r_trg, a_src, a_trg, v_src, v_trg, dr_src, da_src);
            }
        }
    }
}

__global__ void rk4_process(const double dt, double* x, const double* k, const double* d_states, const double factor, const int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < len) {
        x[tid] = k[tid];
        x[tid] *= factor * dt;
        x[tid] += d_states[tid];
    }
}

__global__ void rk4_final(const double dt, double* d_states, double* k1, double* k2, const double* k3, const double* k4, const int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < len) {
        k1[tid] += k4[tid];
        k1[tid] *= dt / 6.;

        k2[tid] += k3[tid];
        k2[tid] *= dt / 3.;

        k1[tid] += k2[tid];

        d_states[tid] += k1[tid];
    }
}

__global__ void scale(double* rates, const double dt, const int len) {
    for (int i = 0; i < len; i++) {
        rates[i] *= dt;
    }
}

__global__ void add(double* states, double* rates, const int len) {
    for (int i = 0; i < len; i++) {
        states[i] += rates[i];
    }
}

void step_cuda(const double dt, pawan::wake_cuda* w, double* d_states, double* rates, const int len) {
    int numBlocks_states = (len / 2 + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlocks_interact = (w->numParticles + BLOCKSIZE - 1) / BLOCKSIZE;

    clear<<<1, 1>>>(*w);
    interact_cuda<<<numBlocks_interact, BLOCKSIZE>>>(*w);
    getRates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, rates);
    scale<<<1, 1>>>(rates, dt, len);
    add<<<1, 1>>>(d_states, rates, len);
    setStates_cuda<<<numBlocks_states, BLOCKSIZE>>>(*w, d_states);
}

extern "C" void cuda_step_wrapper(const double _dt, pawan::wake_struct* w, double* state_array) {
    // TODO: replace hipMallocManaged with hipMalloc for wake_cuda
    pawan::wake_cuda cuda_wake;
    cuda_wake.size = w->size;
    cuda_wake.numParticles = w->numParticles;
    hipMallocManaged(&cuda_wake.position, sizeof(double) * w->size);
    hipMallocManaged(&cuda_wake.velocity, sizeof(double) * w->size);
    hipMallocManaged(&cuda_wake.vorticity, sizeof(double) * w->size);
    hipMallocManaged(&cuda_wake.retvorcity, sizeof(double) * w->size);
    hipMallocManaged(&cuda_wake.radius, sizeof(double) * w->numParticles);
    hipMallocManaged(&cuda_wake.volume, sizeof(double) * w->numParticles);
    hipMallocManaged(&cuda_wake.birthstrength, sizeof(double) * w->numParticles);

    // initialize wake on gpu
    for (size_t i = 0; i < w->numParticles; i++) {
        cuda_wake.radius[i] = w->radius[i];
        cuda_wake.volume[i] = w->volume[i];
        cuda_wake.birthstrength[i] = w->birthstrength[i];
        for (size_t j = 0; j < w->numDimensions; j++) {
            cuda_wake.position[i * w->numDimensions + j] = w->position[i][j];
            cuda_wake.velocity[i * w->numDimensions + j] = w->velocity[i][j];
            cuda_wake.vorticity[i * w->numDimensions + j] = w->vorticity[i][j];
            cuda_wake.retvorcity[i * w->numDimensions + j] = w->retvorcity[i][j];
        }
    }

    // states, ks and xs
    double* d_states;
    hipMalloc(&d_states, sizeof(double) * w->size);
    hipMemcpy(d_states, state_array, sizeof(double) * w->size, hipMemcpyHostToDevice);

    double* rates;
    hipMalloc(&rates, sizeof(double) * w->size);

    double tStart = TIME();
    for (size_t i = 1; i <= STEPS; i++) {
        OUT("\tStep", i);
        step_cuda(_dt, &cuda_wake, d_states, rates, w->size);
    }
    hipDeviceSynchronize();
    double tEnd = TIME();
    OUT("Total Time (s)", tEnd - tStart);

    for (size_t i = 0; i < 20; i++) {
        std::cout << cuda_wake.position[i * 3] << " " << cuda_wake.position[i * 3 + 1] << " " << cuda_wake.position[i * 3 + 2] << " " << std::endl;
    }

    hipFree(d_states);
    hipFree(rates);
    hipFree(cuda_wake.radius);
    hipFree(cuda_wake.volume);
    hipFree(cuda_wake.birthstrength);
    hipFree(cuda_wake.position);
    hipFree(cuda_wake.velocity);
    hipFree(cuda_wake.vorticity);
    hipFree(cuda_wake.retvorcity);
}