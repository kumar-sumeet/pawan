#include "hip/hip_runtime.h"
/*! PArticle Wake ANalysis
 * \file interaction.cpp
 * \brief Routines for interactions
 *
 * @author Puneet Singh
 * @date 04/24/2021
 */
#include "interaction.h"
#include "interaction_utils.h"

pawan::__interaction::__interaction(){
    DOUT("--------------------------------in pawan::__interaction::__interaction()");
    //_nu = 5.0e-3;   //vring fission-fusion
	//_nu = 2.5e-3;   //vring
    _nu = 1.56e-5;    //coupling
    //_nu = 0.0;
	_nWake = 0;
	_totalVorticity = gsl_vector_calloc(3);
	_linearImpulse = gsl_vector_calloc(3);
	_angularImpulse = gsl_vector_calloc(3);
    printf("_nu = %+8.3e\n",_nu);
}

pawan::__interaction::__interaction(__wake *W):__interaction(){
    DOUT("--------------------------------in pawan::__interaction::__interaction(__wake *W):__interaction()");
    //_maxsize = W->_maxsize;
    addWake(W);
}

pawan::__interaction::__interaction(__wake *W1, __wake *W2):__interaction(){
    DOUT("--------------------------------in pawan::__interaction::__interaction(__wake *W1, __wake *W2):__interaction()");
    //_maxsize = 2*W1->_maxsize; //max values same for both wakes
    addWake(W1);
	addWake(W2);
}

pawan::__interaction::~__interaction(){
    DOUT("--------------------------------in pawan::__interaction::~__interaction()");
	gsl_vector_free(_totalVorticity);
	gsl_vector_free(_linearImpulse);
	gsl_vector_free(_angularImpulse);
}

void pawan::__interaction::addWake(__wake *W){
    DOUT("--------------------------------in pawan::__interaction::addWake()");
	_W.push_back(W);
	_size = W->_size; //temp fix for Vring
    _totalmaxsize+= W->_maxsize;
	_nWake++;
    //printf("Position of 1st particle     interaction,addWake(): %+8.3e, %+8.3e, %+8.3e\n",
    //       gsl_matrix_get(W->_position, 0, 0),gsl_matrix_get(W->_position, 0, 1),gsl_matrix_get(W->_position, 0, 2));
}

void pawan::__interaction::solve(){
    DOUT("--------------------------------in pawan::__interaction::solve()");
	interact();
}

void pawan::__interaction::resolve(){
    DOUT("--------------------------------in pawan::__interaction::resolve()");
	influence();
}

void pawan::__interaction::diagnose(){
    DOUT("--------------------------------in pawan::__interaction::diagnose()");
	// Linear diagnostics
	gsl_vector_set_zero(_totalVorticity);
	gsl_vector_set_zero(_linearImpulse);
	gsl_vector_set_zero(_angularImpulse);

	gsl_vector *O = gsl_vector_calloc(3);
	gsl_vector *I = gsl_vector_calloc(3);
	gsl_vector *A = gsl_vector_calloc(3);
    for(auto &w : _W){
		// Total vorticity
		calculateTotalVorticity(w,O);
        gsl_vector_add(_totalVorticity,O);
		// Linear impulse
		calculateLinearImpulse(w,I);
        gsl_vector_add(_linearImpulse,I);
		// Angular impulse
		calculateAngularImpulse(w,A);
        gsl_vector_add(_angularImpulse,A);
	}
	OUTT("Total Vorticity",_totalVorticity);
	OUTT("Linear Impulse",_linearImpulse);
	OUTT("Angular Impulse",_angularImpulse);
	gsl_vector_free(O);
	gsl_vector_free(I);
	gsl_vector_free(A);

	// Quadratic diagnostics
	_helicity = 0.0;
	for(size_t i = 0; i<_nWake; ++i){
		_helicity += calculateHelicity(_W[i]);
		for(size_t j = i + 1; j < _nWake; ++j){
			_helicity += calculateHelicity(_W[i],_W[j]);
		}
	}
	OUT("Helicity",_helicity);
	_enstrophy = 0.0;
	for(size_t i = 0; i<_nWake; ++i){
		_enstrophy += calculateEnstrophy(_W[i]);
		for(size_t j = i + 1; j < _nWake; ++j){
			_enstrophy += calculateEnstrophy(_W[i],_W[j]);
		}
	}
	OUT("Enstrophy",_enstrophy);
    _enstrophyF = 0.0;
    for(size_t i = 0; i<_nWake; ++i){
        _enstrophyF += calculateEnstrophyF(_W[i]);
        for(size_t j = i + 1; j < _nWake; ++j){
            _enstrophyF += calculateEnstrophyF(_W[i],_W[j]);
        }
    }
    OUT("EnstrophyF",_enstrophyF);
    _kineticEnergy = 0.0;
    for(size_t i = 0; i<_nWake; ++i){
        _kineticEnergy += calculateKineticEnergy(_W[i]);
        for(size_t j = i + 1; j < _nWake; ++j){
            _kineticEnergy += calculateKineticEnergy(_W[i],_W[j]);
        }
    }
    OUT("KineticEnergy",_kineticEnergy);
    _kineticEnergyF = 0.0;
    for(size_t i = 0; i<_nWake; ++i){
        _kineticEnergyF += calculateKineticEnergyF(_W[i]);
        for(size_t j = i + 1; j < _nWake; ++j){
            _kineticEnergyF += calculateKineticEnergyF(_W[i],_W[j]);
        }
    }
    OUT("KineticEnergyF",_kineticEnergyF);


    double x = 0.0;
    double y = 0.0;
    double z = 0.0;
    double oy = 0.0;
    for(auto &w : _W) {
        double num = 0.0;
        double denom = 0.0;
        for (size_t i = 0; i < w->_numParticles; ++i) {
            gsl_vector_const_view ivor = gsl_matrix_const_row(w->_vorticity, i);
            oy = gsl_blas_dnrm2(&ivor.vector);
            x = gsl_matrix_get(w->_position, i, 0);
            y = gsl_matrix_get(w->_position, i, 1);
            z = gsl_matrix_get(w->_position, i, 2);
            num += oy * z * (x * x + y * y);
            denom += oy * (x * x + y * y);
            //num += oy * z;
            //denom += oy;
        }
        if (denom != 0.0) {
            _Zc = num / denom;
        }
        OUT("num ",num);
        OUT("denom ",denom);
    }
    OUT("Centroid of wake Zc",_Zc);

/* This is leading to a wrong result (WHY???)
    gsl_vector *rxo = gsl_vector_calloc(3);
    gsl_vector *Zc = gsl_vector_calloc(3);
    gsl_vector *dZc = gsl_vector_calloc(3);
    double rxoI = 0.0;
    double li = gsl_blas_dnrm2(_linearImpulse);
    for(auto &w : _W) {
        for (size_t i = 0; i < w->_numParticles; ++i) {
            gsl_vector_const_view ipos = gsl_matrix_const_row(w->_position, i);
            gsl_vector_const_view ivor = gsl_matrix_const_row(w->_vorticity, i);
            gsl_cross(&ipos.vector, &ivor.vector, rxo);
            gsl_blas_ddot(rxo, I, &rxoI);
            gsl_vector_memcpy(dZc, &ipos.vector);
            if (li != 0.0) {
                gsl_blas_dscal(0.5 * rxoI / li / li, dZc);
                gsl_vector_add(Zc, dZc);
            }
        }
        OUTT("Centroid of wake Zc", Zc);
        gsl_vector_free(rxo);
        gsl_vector_free(dZc);
        gsl_vector_free(Zc);
    }
*/

//vel at origin of ring
gsl_vector *vbi = gsl_vector_calloc(3);
gsl_vector *r = gsl_vector_calloc(3);
for (size_t k = 0; k < 3; ++k) {
    gsl_vector_set(r, k, 0.0);
}
getVi(r,vbi,0);
OUTT("vbi",vbi);
gsl_vector_free(vbi);
gsl_vector_free(r);
}

void pawan::__interaction::interact(){
    DOUT("--------------------------------in pawan::__interaction::interact()");
	for(auto &w : _W){
		gsl_matrix_set_zero(w->_velocity);
		gsl_matrix_set_zero(w->_retvorcity);
	}
	for(auto &w : _W){
		interact(w);
	}
	for(size_t i = 0; i<_nWake; ++i){
		for(size_t j = i + 1; j < _nWake; ++j){
			interact(_W[i],_W[j]);
		}
	}
}

void pawan::__interaction::influence(){
    DOUT("--------------------------------in pawan::__interaction::influence()");
	for(auto &w : _W){
		gsl_matrix_set_zero(w->_vorticityfield);
	}
	for(auto &w : _W){
		influence(w);
	}
	for(size_t i = 0; i<_nWake; ++i){
		for(size_t j = i + 1; j < _nWake; ++j){
			influence(_W[i],_W[j]);
		}
	}
}

void pawan::__interaction::influence(__wake *W){
    DOUT("--------------------------------in pawan::__interaction::influence(__wake *W)");
	for(size_t i_src = 0; i_src < W->_numParticles; ++i_src){
		gsl_vector_const_view r_src = gsl_matrix_const_row(W->_position,i_src);
		gsl_vector_const_view a_src = gsl_matrix_const_row(W->_vorticity,i_src);
		gsl_vector_view k_src = gsl_matrix_row(W->_vorticityfield,i_src);
		double s_src = gsl_vector_get(W->_radius,i_src);
		SELFINFLUENCE(s_src,&r_src.vector,&a_src.vector,&k_src.vector);
		for(size_t i_trg = i_src + 1; i_trg < W->_numParticles; ++i_trg){
			gsl_vector_const_view r_trg = gsl_matrix_const_row(W->_position,i_trg);
			gsl_vector_const_view a_trg= gsl_matrix_const_row(W->_vorticity,i_trg);
			gsl_vector_view k_trg = gsl_matrix_row(W->_vorticityfield,i_trg);
			double s_trg = gsl_vector_get(W->_radius,i_trg);
			INFLUENCE(s_src,s_trg,&r_src.vector,&r_trg.vector,&a_src.vector,&a_trg.vector,&k_src.vector,&k_trg.vector);
		}
	}
}

void pawan::__interaction::influence(__wake *W1, __wake *W2){
    DOUT("--------------------------------in pawan::__interaction::influence(__wake *W1, __wake *W2)");
	for(size_t i_src = 0; i_src < W1->_numParticles; ++i_src){
		gsl_vector_const_view r_src = gsl_matrix_const_row(W1->_position,i_src);
		gsl_vector_const_view a_src = gsl_matrix_const_row(W1->_vorticity,i_src);
		gsl_vector_view k_src = gsl_matrix_row(W1->_vorticityfield,i_src);
		double s_src = gsl_vector_get(W1->_radius,i_src);
		for(size_t i_trg = 0; i_trg < W2->_numParticles; ++i_trg){
			gsl_vector_const_view r_trg = gsl_matrix_const_row(W2->_position,i_trg);
			gsl_vector_const_view a_trg = gsl_matrix_const_row(W2->_vorticity,i_trg);
			gsl_vector_view k_trg = gsl_matrix_row(W2->_vorticityfield,i_trg);
			double s_trg = gsl_vector_get(W2->_radius,i_trg);
			INFLUENCE(s_src,s_trg,&r_src.vector,&r_trg.vector,&a_src.vector,&a_trg.vector,&k_src.vector,&k_trg.vector);
		}
	}
}

void pawan::__interaction::interact(__wake *W){
    DOUT("--------------------------------in pawan::__interaction::interact(__wake *W)");
	for(size_t i_src = 0; i_src < W->_numParticles; ++i_src){
		gsl_vector_const_view r_src = gsl_matrix_const_row(W->_position,i_src);
		gsl_vector_const_view a_src = gsl_matrix_const_row(W->_vorticity,i_src);
		gsl_vector_view dr_src = gsl_matrix_row(W->_velocity,i_src);
		gsl_vector_view da_src = gsl_matrix_row(W->_retvorcity,i_src);
		double s_src = gsl_vector_get(W->_radius,i_src);
		double v_src = gsl_vector_get(W->_volume,i_src);
		for(size_t i_trg = i_src + 1; i_trg < W->_numParticles; ++i_trg){
			gsl_vector_const_view r_trg = gsl_matrix_const_row(W->_position,i_trg);
			gsl_vector_const_view a_trg= gsl_matrix_const_row(W->_vorticity,i_trg);
			gsl_vector_view dr_trg = gsl_matrix_row(W->_velocity,i_trg);
			gsl_vector_view da_trg = gsl_matrix_row(W->_retvorcity,i_trg);
			double s_trg = gsl_vector_get(W->_radius,i_trg);
			double v_trg = gsl_vector_get(W->_volume,i_trg);
			INTERACT(_nu,s_src,s_trg,&r_src.vector,&r_trg.vector,&a_src.vector,&a_trg.vector,v_src,v_trg,&dr_src.vector,&dr_trg.vector,&da_src.vector,&da_trg.vector);
		}
	}
}

void pawan::__interaction::interact(__wake *W1, __wake *W2){
    DOUT("--------------------------------in pawan::__interaction::interact(__wake *W1, __wake *W2)");
	for(size_t i_src = 0; i_src < W1->_numParticles; ++i_src){
		gsl_vector_const_view r_src = gsl_matrix_const_row(W1->_position,i_src);
		gsl_vector_const_view a_src = gsl_matrix_const_row(W1->_vorticity,i_src);
		gsl_vector_view dr_src = gsl_matrix_row(W1->_velocity,i_src);
		gsl_vector_view da_src = gsl_matrix_row(W1->_retvorcity,i_src);
		double s_src = gsl_vector_get(W1->_radius,i_src);
		double v_src = gsl_vector_get(W1->_volume,i_src);
		for(size_t i_trg = 0; i_trg < W2->_numParticles; ++i_trg){
			gsl_vector_const_view r_trg = gsl_matrix_const_row(W2->_position,i_trg);
			gsl_vector_const_view a_trg = gsl_matrix_const_row(W2->_vorticity,i_trg);
			gsl_vector_view dr_trg = gsl_matrix_row(W2->_velocity,i_trg);
			gsl_vector_view da_trg = gsl_matrix_row(W2->_retvorcity,i_trg);
			double s_trg = gsl_vector_get(W2->_radius,i_trg);
			double v_trg = gsl_vector_get(W2->_volume,i_trg);
			INTERACT(_nu,s_src,s_trg,&r_src.vector,&r_trg.vector,&a_src.vector,&a_trg.vector,v_src,v_trg,&dr_src.vector,&dr_trg.vector,&da_src.vector,&da_trg.vector);
		}
	}
}

void pawan::__interaction::write(FILE *f){
    DOUT("--------------------------------in pawan::__interaction::write(FILE *f)");
	fwrite(&_nWake,sizeof(size_t),1,f);

    int write = 1;
	for(auto &w: _W){
	    if (write){ //max particles same for all wake, write only once
            int maxnumparticles = w->_maxnumParticles;
            fwrite(&maxnumparticles,sizeof(size_t),1,f);
	        write=0;
	    }
        //int maxnumparticles = w->_maxnumParticles;
        //fwrite(&maxnumparticles,sizeof(size_t),1,f);
		w->write(f);
	}
}

void pawan::__interaction::writenu(FILE *fdiag){
    fwrite(&_nu,sizeof(double),1,fdiag);
}

void pawan::__interaction::setDiagnostics(double *totalDiag){
    for(size_t i = 0; i<3; ++i) {
        gsl_vector_set(_totalVorticity, i, totalDiag[i]);
        gsl_vector_set(_linearImpulse, i, totalDiag[3+i]);
        gsl_vector_set(_angularImpulse, i, totalDiag[6+i]);
    }
    _enstrophy = totalDiag[9];
    _kineticEnergy = totalDiag[10];
    _helicity = totalDiag[11];
    _enstrophyF = totalDiag[12];
    _kineticEnergyF = totalDiag[13];
    if(totalDiag[15]!=0)
        _Zc = totalDiag[14]/totalDiag[15];
}

void pawan::__interaction::writediagnosis(FILE *fdiag){
    gsl_vector_fwrite(fdiag,_totalVorticity);
    gsl_vector_fwrite(fdiag,_linearImpulse);
    gsl_vector_fwrite(fdiag,_angularImpulse);
    fwrite(&_helicity,sizeof(double),1,fdiag);
    fwrite(&_enstrophy,sizeof(double),1,fdiag);
    fwrite(&_enstrophyF,sizeof(double),1,fdiag);
    fwrite(&_kineticEnergy,sizeof(double),1,fdiag);
    fwrite(&_kineticEnergyF,sizeof(double),1,fdiag);
    fwrite(&_Zc,sizeof(double),1,fdiag);
}

void pawan::__interaction::printdiagnostics(double *totalDiag){
    printf("\tTotal Vorticity = \t%21.16e %21.16e %21.16e \n", totalDiag[0], totalDiag[1], totalDiag[2]);
    printf("\tLinear Impulse  = \t%21.16e %21.16e %21.16e \n", totalDiag[3], totalDiag[4], totalDiag[5]);
    printf("\tAngular Impulse = \t%21.16e %21.16e %21.16e \n", totalDiag[6], totalDiag[7], totalDiag[8]);
    printf("Helicity            = %21.16e \n", totalDiag[11]);
    printf("Enstrophy           = %21.16e \n", totalDiag[9]);
    printf("EnstrophyF          = %21.16e \n", totalDiag[12]);
    printf("KineticEnergy       = %21.16e \n", totalDiag[10]);
    printf("KineticEnergyF      = %21.16e \n", totalDiag[13]);
    if(totalDiag[15]!=0) {
        printf("Centroid of wake Zc = %21.16e \n", totalDiag[14] / totalDiag[15]);
    }
    printf("	vbi = %21.16e, %21.16e, %21.16e \n", totalDiag[17],totalDiag[18],totalDiag[19]);
}

void pawan::__interaction::setStates(const gsl_vector *state){
    DOUT("--------------------------------in pawan::__interaction::setStates(const gsl_vector *state)");
	size_t offset = 0;
	for(auto &w: _W){
		gsl_vector_const_view st = gsl_vector_const_subvector(state,offset,w->_maxsize);
		w->setStates(&st.vector);
		offset += w->_maxsize;
	}
}

void pawan::__interaction::getRates(gsl_vector *rate){
    DOUT("--------------------------------in pawan::__interaction::getRates(gsl_vector *rate)");
	size_t offset = 0;
	for(auto &w: _W){
		gsl_vector_view rt = gsl_vector_subvector(rate,offset,w->_maxsize);
		w->getRates(&rt.vector);
		offset += w->_maxsize;
	}
}

void pawan::__interaction::getStates(gsl_vector *state){
    DOUT("--------------------------------in pawan::__interaction::getStates(gsl_vector *state)");
	size_t offset = 0;
	for(auto &w: _W){
		gsl_vector_view st = gsl_vector_subvector(state,offset,w->_maxsize);
		w->getStates(&st.vector);
		offset += w->_maxsize;
	}
}

void pawan::__interaction::split(size_t &stepnum){
    size_t offset = 0;
    for(auto &w: _W){
        w->split(stepnum);
        offset += w->_maxsize;
    }
}

void pawan::__interaction::merge(size_t &stepnum){
    size_t offset = 0;
    for(auto &w: _W){
        w->merge(stepnum);
        offset += w->_maxsize;
    }
}

void pawan::__interaction::addParticles(PawanRecvData pawanrecvdata,size_t &stepnum){
    size_t offset = 0;
    for(auto &w: _W){
        w->addParticles(pawanrecvdata,stepnum);
        offset += w->_maxsize;
    }
}

void pawan::__interaction::updateVinfEffect(const double *Vinf,double &dt){
    size_t offset = 0;
    for(auto &w: _W){
        w->updateVinfEffect(Vinf,dt);
        offset += w->_maxsize;
    }
}

void pawan::__interaction::updateBoundVorBoundVorEffectVind(PawanRecvData pawanrecvdata,PawanSendData pawansenddata,int astidx, int lfnidx){
    for(auto &w: _W){
        w->updateBoundVorBoundVorEffectVind(pawanrecvdata,pawansenddata,astidx, lfnidx);
    }
}
void pawan::__interaction::updateBoundVorEffect(PawanRecvData pawanrecvdata,double &dt,size_t &stepnum){
    size_t offset = 0;
    for(auto &w: _W){
        w->updateBoundVorEffect(pawanrecvdata,dt,stepnum); //effect of all lifting surfaces on each wake
        offset += w->_maxsize;
    }
}

void pawan::__interaction::getInflow(PawanRecvData pawanrecvdata, PawanSendData pawansenddata){
    int NbOfLfnLines = pawanrecvdata->NbOfLfnLines;
    int *NbOfAst = pawanrecvdata->NbOfAst;
    double *astpos = pawanrecvdata->astpos;

    double *lambda = pawansenddata->lambda;
    for (size_t j=0; j<PAWAN_MAXLFNLINES*PAWAN_MAXAST*3; j++){
        lambda[j]=0.0;
    }
    int astidx = 0;
    for (size_t ilfn = 0; ilfn < NbOfLfnLines; ++ilfn) {
        printf("------------------------------\n");
        for (size_t iast = 0; iast < NbOfAst[ilfn]; ++iast) {
            gsl_vector *vbi = gsl_vector_calloc(3);
            gsl_vector *rast = gsl_vector_calloc(3);
            for (size_t k = 0; k < 3; ++k) {
                gsl_vector_set(rast, k, astpos[astidx*3 + k]);
            }

            getVi(rast,vbi,iast);

            for (size_t k = 0; k < 3; ++k) {
                lambda[astidx*3 + k] = gsl_vector_get(vbi, k);
            }
/*            printf("---> lambda = %+10.5e, %+10.5e, %+10.5e @ast = %+10.5e, %+10.5e, %+10.5e  \n",
                   lambda[astidx*3],lambda[astidx*3 + 1],lambda[astidx*3 + 2],
                   astpos[astidx*3],astpos[astidx*3 + 1],astpos[astidx*3 + 2]);
*/          printf("CPU---> lambda = %+10.5e, %+10.5e, %+10.5e \n",
            lambda[astidx*3],lambda[astidx*3 + 1],lambda[astidx*3 + 2]);
            astidx++;
            gsl_vector_free(vbi);
            gsl_vector_free(rast);
        }
    }

    gsl_vector *vi = gsl_vector_calloc(3);
    gsl_vector *r = gsl_vector_calloc(3);
    gsl_vector_set(r,0,0);gsl_vector_set(r,1,0.8);gsl_vector_set(r,2,0);
    gsl_vector_set_zero(vi);
    getVi(r,vi,0);
    printf("Vi at Mid-wing ast = %10.5e, %10.5e, %10.5e \n",gsl_vector_get(vi,0),gsl_vector_get(vi,1),gsl_vector_get(vi,2));
    gsl_vector_set(r,0,-0.255);gsl_vector_set(r,1,0.8);gsl_vector_set(r,2,0.02);
    gsl_vector_set_zero(vi);
    getVi(r,vi,0);
    printf("Vi at Mid-wing TE  = %10.5e, %10.5e, %10.5e \n",gsl_vector_get(vi,0),gsl_vector_get(vi,1),gsl_vector_get(vi,2));
    gsl_vector_set(r,0,-2.5);gsl_vector_set(r,1,0.8);gsl_vector_set(r,2,0.02);
    gsl_vector_set_zero(vi);
    getVi(r,vi,0);
    printf("Vi at point 2 = %10.5e, %10.5e, %10.5e \n",gsl_vector_get(vi,0),gsl_vector_get(vi,1),gsl_vector_get(vi,2));
    gsl_vector_set(r,0,0.0);gsl_vector_set(r,1,1.6);gsl_vector_set(r,2,0.0);
    gsl_vector_set_zero(vi);
    getVi(r,vi,0);
    printf("Vi at point 3 = %10.5e, %10.5e, %10.5e \n",gsl_vector_get(vi,0),gsl_vector_get(vi,1),gsl_vector_get(vi,2));

    gsl_vector_free(vi);gsl_vector_free(r);

}

void pawan::__interaction::getVi(const gsl_vector *r, gsl_vector *vi, const size_t &n){
    for (auto &W: _W) {//induced inflow due to each wake
        for (size_t i = 0; i < W->_numParticles; ++i) {
            gsl_vector *displacement = gsl_vector_calloc(3);
            gsl_vector_const_view ipos = gsl_matrix_const_row(W->_position, i);
            gsl_vector_const_view ivor = gsl_matrix_const_row(W->_vorticity, i);
            double sigma = gsl_vector_get(W->_radius, i);

            gsl_vector_memcpy(displacement, r);
            gsl_vector_sub(displacement, &ipos.vector);
            double rho = gsl_blas_dnrm2(displacement);
            double q = 0.0;
            q = QSIG(rho,sigma);
            //if(i%17==0 && n==8)
                //printf("q = %10.5e \t",q);
            // Velocity computation
            gsl_vector *dv = gsl_vector_calloc(3);
            VELOCITY(-q, &ivor.vector, displacement, dv);
            gsl_vector_add(vi, dv);

            gsl_vector_free(displacement);
            gsl_vector_free(dv);
        }
    }
}

void pawan::__interaction::getIdealRates(gsl_vector *rate){
    DOUT("--------------------------------in pawan::__interaction::getIdealRates(gsl_vector *rate)");
	size_t offset = 0;
	for(auto &w: _W){
		gsl_vector_view rt = gsl_vector_subvector(rate,offset,w->_maxsize);
		w->getIdealRates(&rt.vector);
		offset += w->_maxsize;
	}
}

void pawan::__interaction::calculateTotalVorticity(__wake *W, gsl_vector *O){
    DOUT("--------------------------------in pawan::__interaction::calculateTotalVorticity(__wake *W, gsl_vector *O)");
	gsl_vector_set_zero(O); //ip: redundant, *O was already set as calloc
	for(size_t i = 0; i < W->_numParticles; ++i){
		gsl_vector_const_view a = gsl_matrix_const_row(W->_vorticity,i);
		gsl_vector_add(O,&a.vector);
	}
}

void pawan::__interaction::calculateLinearImpulse(__wake *W, gsl_vector *I){
    DOUT("--------------------------------in pawan::__interaction::calculateLinearImpulse(__wake *W, gsl_vector *I)");
	gsl_vector_set_zero(I);//ip: redundant
	gsl_vector *rxa = gsl_vector_calloc(3);
	for(size_t i = 0; i < W->_numParticles; ++i){
		gsl_vector_const_view r = gsl_matrix_const_row(W->_position,i);
		gsl_vector_const_view a = gsl_matrix_const_row(W->_vorticity,i);
		gsl_cross(&r.vector,&a.vector,rxa);
		gsl_vector_add(I,rxa);
	}
	gsl_vector_scale(I,0.5);
	gsl_vector_free(rxa);
}

void pawan::__interaction::calculateAngularImpulse(__wake *W, gsl_vector *A){
    DOUT("--------------------------------in pawan::__interaction::calculateAngularImpulse(__wake *W, gsl_vector *A)");
	gsl_vector_set_zero(A);//ip: redundant
	gsl_vector *rxa = gsl_vector_calloc(3);
	gsl_vector *rxrxa = gsl_vector_calloc(3);
	for(size_t i = 0; i < W->_numParticles; ++i){
		gsl_vector_const_view r = gsl_matrix_const_row(W->_position,i);
		gsl_vector_const_view a = gsl_matrix_const_row(W->_vorticity,i);
		gsl_cross(&r.vector,&a.vector,rxa);
		gsl_cross(&r.vector,rxa,rxrxa);
		gsl_vector_add(A,rxrxa);
	}
	gsl_vector_scale(A,1.0/3.0);
	gsl_vector_free(rxrxa);
	gsl_vector_free(rxa);
}

double pawan::__interaction::calculateEnstrophy(__wake *W){
    DOUT("--------------------------------in pawan::__interaction::calculateEnstrophy(__wake *W)");
	double s = 0.0;
	for(size_t I = 0; I < W->_numParticles; ++I){
		gsl_vector_const_view rI = gsl_matrix_const_row(W->_position,I);
		gsl_vector_const_view aI = gsl_matrix_const_row(W->_vorticity,I);
		double sI = gsl_vector_get(W->_radius,I);
		s += ENSTROPHY(sI,&aI.vector);
		//double ens = ENSTROPHY(sI,&aI.vector);
		//s += ens;
		//OUT("e",ens);
		for(size_t J = I + 1; J < W->_numParticles; ++J){
			//OUT("\tJ",J);
			gsl_vector_const_view rJ = gsl_matrix_const_row(W->_position,J);
			gsl_vector_const_view aJ = gsl_matrix_const_row(W->_vorticity,J);
			double sJ = gsl_vector_get(W->_radius,J);
			// doubling S because S(i,j) = S(j,i)
			s += 2*ENSTROPHY(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
			//OUT("\ts",s);
		}
	}
	return s;
}

double pawan::__interaction::calculateEnstrophy(__wake *W1, __wake *W2){
	double s = 0.0;
	for(size_t I = 0; I < W1->_numParticles; ++I){
		gsl_vector_const_view rI = gsl_matrix_const_row(W1->_position,I);
		gsl_vector_const_view aI = gsl_matrix_const_row(W1->_vorticity,I);
		double sI = gsl_vector_get(W1->_radius,I);
		for(size_t J = 0; J < W2->_numParticles; ++J){
			gsl_vector_const_view rJ = gsl_matrix_const_row(W2->_position,J);
			gsl_vector_const_view aJ = gsl_matrix_const_row(W2->_vorticity,J);
			double sJ = gsl_vector_get(W2->_radius,J);
			s += 2.0*ENSTROPHY(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
		}
	}
	// doubling S because S(i,j) = S(j,i)
	return s;
}

double pawan::__interaction::calculateEnstrophyF(__wake *W){
    DOUT("--------------------------------in pawan::__interaction::calculateEnstrophyF(__wake *W)");
    double s = 0.0;
    for(size_t I = 0; I < W->_numParticles; ++I){
        gsl_vector_const_view rI = gsl_matrix_const_row(W->_position,I);
        gsl_vector_const_view aI = gsl_matrix_const_row(W->_vorticity,I);
        double sI = gsl_vector_get(W->_radius,I);
        s += ENSTROPHYF(sI,&aI.vector);
        for(size_t J = I + 1; J < W->_numParticles; ++J){
            //OUT("\tJ",J);
            gsl_vector_const_view rJ = gsl_matrix_const_row(W->_position,J);
            gsl_vector_const_view aJ = gsl_matrix_const_row(W->_vorticity,J);
            double sJ = gsl_vector_get(W->_radius,J);
            // doubling S because S(i,j) = S(j,i)
            s += 2*ENSTROPHYF(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
            //OUT("\ts",s);
        }
    }
    return s;
}

double pawan::__interaction::calculateEnstrophyF(__wake *W1, __wake *W2){
    double s = 0.0;
    for(size_t I = 0; I < W1->_numParticles; ++I){
        gsl_vector_const_view rI = gsl_matrix_const_row(W1->_position,I);
        gsl_vector_const_view aI = gsl_matrix_const_row(W1->_vorticity,I);
        double sI = gsl_vector_get(W1->_radius,I);
        for(size_t J = 0; J < W2->_numParticles; ++J){
            gsl_vector_const_view rJ = gsl_matrix_const_row(W2->_position,J);
            gsl_vector_const_view aJ = gsl_matrix_const_row(W2->_vorticity,J);
            double sJ = gsl_vector_get(W2->_radius,J);
            s += 2.0*ENSTROPHYF(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
        }
    }
    // doubling S because S(i,j) = S(j,i)
    return s;
}

double pawan::__interaction::calculateHelicity(__wake *W){
    DOUT("--------------------------------in pawan::__interaction::calculateHelicity(__wake *W)");
	int test_counter=0;
    double h = 0.0;
	for(size_t I = 0; I < W->_numParticles; ++I){
		gsl_vector_const_view rI = gsl_matrix_const_row(W->_position,I);
		gsl_vector_const_view aI = gsl_matrix_const_row(W->_vorticity,I);
		double sI = gsl_vector_get(W->_radius,I);
		for(size_t J = I + 1; J < W->_numParticles; ++J){
			gsl_vector_const_view rJ = gsl_matrix_const_row(W->_position,J);
			gsl_vector_const_view aJ = gsl_matrix_const_row(W->_vorticity,J);
			double sJ = gsl_vector_get(W->_radius,J);
            test_counter = test_counter+1;
			//printf("I=%d, J=%d ==== %d",I,J,test_counter);
            //printf("number of particles %d",W->_numParticles);
            h += 2.0*HELICITY(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
		}
	}
	// doubling H because H(i,j) = H(j,i)
	return h;
}

double pawan::__interaction::calculateHelicity(__wake *W1, __wake *W2){
    DOUT("--------------------------------in pawan::__interaction::calculateHelicity(__wake *W1, __wake *W2)");
	double h = 0.0;
	for(size_t I = 0; I < W1->_numParticles; ++I){
		gsl_vector_const_view rI = gsl_matrix_const_row(W1->_position,I);
		gsl_vector_const_view aI = gsl_matrix_const_row(W1->_vorticity,I);
		double sI = gsl_vector_get(W1->_radius,I);
		for(size_t J = 0; J < W2->_numParticles; ++J){
			gsl_vector_const_view rJ = gsl_matrix_const_row(W2->_position,J);
			gsl_vector_const_view aJ = gsl_matrix_const_row(W2->_vorticity,J);
			double sJ = gsl_vector_get(W2->_radius,J);
			h += 2.0*HELICITY(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
		}
	}
	// doubling H because H(i,j) = H(j,i)
	return h;
}

double pawan::__interaction::calculateKineticEnergy(__wake *W){
    DOUT("--------------------------------in pawan::__interaction::calculateKineticEnergy(__wake *W)");
	double ke = 0.0;
	for(size_t I = 0; I < W->_numParticles; ++I){
		gsl_vector_const_view rI = gsl_matrix_const_row(W->_position,I);
		gsl_vector_const_view aI = gsl_matrix_const_row(W->_vorticity,I);
		double sI = gsl_vector_get(W->_radius,I);
		ke += KINETICENERGY(sI,&aI.vector);
		for(size_t J = I + 1; J < W->_numParticles; ++J){
			gsl_vector_const_view rJ = gsl_matrix_const_row(W->_position,J);
			gsl_vector_const_view aJ = gsl_matrix_const_row(W->_vorticity,J);
			double sJ = gsl_vector_get(W->_radius,J);
			ke += 2.0*KINETICENERGY(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
		}
	}
	return ke;
}

double pawan::__interaction::calculateKineticEnergy(__wake *W1, __wake *W2){
    DOUT("--------------------------------in pawan::__interaction::calculateKineticEnergy(__wake *W1, __wake *W2)");
	double ke = 0.0;
	for(size_t I = 0; I < W1->_numParticles; ++I){
		gsl_vector_const_view rI = gsl_matrix_const_row(W1->_position,I);
		gsl_vector_const_view aI = gsl_matrix_const_row(W1->_vorticity,I);
		double sI = gsl_vector_get(W1->_radius,I);
		for(size_t J = 0; J < W2->_numParticles; ++J){
			gsl_vector_const_view rJ = gsl_matrix_const_row(W2->_position,J);
			gsl_vector_const_view aJ = gsl_matrix_const_row(W2->_vorticity,J);
			double sJ = gsl_vector_get(W2->_radius,J);
			ke += 2.0*KINETICENERGY(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
		}
	}
	// doubling KE because KE(i,j) = KE(j,i)
	return ke;
}

double pawan::__interaction::calculateKineticEnergyF(__wake *W){
    DOUT("--------------------------------in pawan::__interaction::calculateKineticEnergy(__wake *W)");
    double ke = 0.0;
    for(size_t I = 0; I < W->_numParticles; ++I){
        gsl_vector_const_view rI = gsl_matrix_const_row(W->_position,I);
        gsl_vector_const_view aI = gsl_matrix_const_row(W->_vorticity,I);
        double sI = gsl_vector_get(W->_radius,I);
        ke += KINETICENERGYF(sI,&aI.vector);
        for(size_t J = I + 1; J < W->_numParticles; ++J){
            gsl_vector_const_view rJ = gsl_matrix_const_row(W->_position,J);
            gsl_vector_const_view aJ = gsl_matrix_const_row(W->_vorticity,J);
            double sJ = gsl_vector_get(W->_radius,J);
            ke += 2.0*KINETICENERGYF(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
        }
    }
    return ke;
}

double pawan::__interaction::calculateKineticEnergyF(__wake *W1, __wake *W2){
    DOUT("--------------------------------in pawan::__interaction::calculateKineticEnergy(__wake *W1, __wake *W2)");
    double ke = 0.0;
    for(size_t I = 0; I < W1->_numParticles; ++I){
        gsl_vector_const_view rI = gsl_matrix_const_row(W1->_position,I);
        gsl_vector_const_view aI = gsl_matrix_const_row(W1->_vorticity,I);
        double sI = gsl_vector_get(W1->_radius,I);
        for(size_t J = 0; J < W2->_numParticles; ++J){
            gsl_vector_const_view rJ = gsl_matrix_const_row(W2->_position,J);
            gsl_vector_const_view aJ = gsl_matrix_const_row(W2->_vorticity,J);
            double sJ = gsl_vector_get(W2->_radius,J);
            ke += 2.0*KINETICENERGYF(sI,sJ,&rI.vector,&rJ.vector,&aI.vector,&aJ.vector);
        }
    }
    // doubling KE because KE(i,j) = KE(j,i)
    return ke;
}

void pawan::__interaction::setParticles(double *p){
    auto rateBuffer = reinterpret_cast<double4*>(p);

    int index = 0;

    for(auto w : _W){
        for(int i = 0; i < w->_numParticles; i++, index++){
            gsl_matrix_set(w->_position,i,0,rateBuffer[2 * index].x);
            gsl_matrix_set(w->_position,i,1,rateBuffer[2 * index].y);
            gsl_matrix_set(w->_position,i,2,rateBuffer[2 * index].z);
            gsl_matrix_set(w->_vorticity,i,0,rateBuffer[2 * index + 1].x);
            gsl_matrix_set(w->_vorticity,i,1,rateBuffer[2 * index + 1].y);
            gsl_matrix_set(w->_vorticity,i,2,rateBuffer[2 * index + 1].z);
            //ignore volume and smoothing radius because they should not have changed
        }
    }

}

void pawan::__interaction::getParticles(double *p,int *age, size_t stepnum){

    auto particlesBuffer = reinterpret_cast<double4*>(p);

    int position = 0;

    for(auto const w : _W){
        for(int i = 0; i < w->_numParticles; i++, position++) {
            particlesBuffer[2 * position].x = gsl_matrix_get(w->_position, i, 0);
            particlesBuffer[2 * position].y = gsl_matrix_get(w->_position, i, 1);
            particlesBuffer[2 * position].z = gsl_matrix_get(w->_position, i, 2);
            particlesBuffer[2 * position].w = gsl_vector_get(w->_radius, i);
            particlesBuffer[2 * position + 1].x = gsl_matrix_get(w->_vorticity, i, 0);
            particlesBuffer[2 * position + 1].y = gsl_matrix_get(w->_vorticity, i, 1);
            particlesBuffer[2 * position + 1].z = gsl_matrix_get(w->_vorticity, i, 2);
            particlesBuffer[2 * position + 1].w = gsl_vector_get(w->_volume, i);
            age[position] = stepnum - gsl_vector_get(w->_active, i);
        }
    }
}

void pawan::__interaction::getParticles_arr(double *p){

    int position = 0;

    int totalparticles = amountParticles();

    for(auto const w : _W){
        for(int i = 0; i < w->_numParticles; i++, position++) {
            p[                   position] = gsl_matrix_get(w->_position, i, 0);
            p[  totalparticles + position] = gsl_matrix_get(w->_position, i, 1);
            p[2*totalparticles + position] = gsl_matrix_get(w->_position, i, 2);
            p[3*totalparticles + position] = gsl_vector_get(w->_radius,   i);
            p[4*totalparticles + position] = gsl_matrix_get(w->_vorticity, i, 0);
            p[5*totalparticles + position] = gsl_matrix_get(w->_vorticity, i, 1);
            p[6*totalparticles + position] = gsl_matrix_get(w->_vorticity, i, 2);
            p[7*totalparticles + position] = gsl_vector_get(w->_volume,    i);
            p[8*totalparticles + position] = sqrt( gsl_matrix_get(w->_vorticity, i, 0)*gsl_matrix_get(w->_vorticity, i, 0)
                                                  +gsl_matrix_get(w->_vorticity, i, 1)*gsl_matrix_get(w->_vorticity, i, 1)
                                                  +gsl_matrix_get(w->_vorticity, i, 2)*gsl_matrix_get(w->_vorticity, i, 2)
                                                 );
        }
    }

}

    int pawan::__interaction::amountParticles() {
    int total = 0;
    for(auto const w : _W){
        total += w->_numParticles;
    }
    return total;
}

int pawan::__interaction::totalmaxParticles() {
    int totalmaxparticles = 0;
    for(auto const w : _W){
        totalmaxparticles += w->_maxnumParticles;
    }
    return totalmaxparticles;
}

double pawan::__interaction::getNu() {
    return _nu;
}
